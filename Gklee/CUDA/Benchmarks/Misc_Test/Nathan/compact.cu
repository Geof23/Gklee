#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>

#define N 4

#define PREDICATE(x) (((x & 1) == 0) ? 1 : 0)

__global__ void compact(int *out, int*in) {
  __shared__ unsigned flag[N];
  __shared__ unsigned idx[N];

  unsigned t = threadIdx.x;

  // (i) test each element with predicate p
  // flag = 1 if keeping element
  //        0 otherwise
  printf("before predicate\n");
  flag[t] = PREDICATE(in[t]);

  // (ii) compute indexes for scatter
  //      using an exclusive prefix sum
  __syncthreads();
  if (t < N/2) {
    idx[2*t]   = flag[2*t];
    idx[2*t+1] = flag[2*t+1];
  }
  // (a) upsweep
  int offset = 1;
  for (unsigned d = N/2; d > 0; d /= 2) {
    __syncthreads();
    if (t < d) {
      int ai = offset * (2 * t + 1) - 1;
      int bi = offset * (2 * t + 2) - 1;
      idx[bi] += idx[ai];
    }
    offset *= 2;
  }
  // (b) downsweep
  if (t == 0) idx[N-1] = 0;
  for (unsigned d = 1; d < N; d *= 2) {
    offset /= 2;
    __syncthreads();
    if (t < d) {
      int ai = offset * (2 * t + 1) - 1;
      int bi = offset * (2 * t + 2) - 1;
      int temp = idx[ai];
      idx[ai] = idx[bi];
      idx[bi] += temp;
    }
  }
  __syncthreads();
  // end of exclusive prefix sum of flag into idx

  // (iii) scatter
  if (flag[t]) out[idx[t]] = in[t];
}

int main(int argc, char **argv) {
  // test data
  size_t ArraySize = N * sizeof(int);
  int *in  = (int *)malloc(ArraySize);
  int *out = (int *)malloc(ArraySize);
  klee_make_symbolic(in, ArraySize, "in");

  // create some memory objects on the device
  int *d_in;
  int *d_out;
  hipMalloc((void **)&d_in, ArraySize);
  hipMalloc((void **)&d_out, ArraySize);

  // memcpy into these objects
  hipMemcpy(d_in, in, ArraySize, hipMemcpyHostToDevice);

  // run the kernel
  compact<<<1,N>>>(d_out, d_in);

  printf("finish GPU mode\n");
  // memcpy back the result
  hipMemcpy(out, d_out, ArraySize, hipMemcpyDeviceToHost);

#ifndef _SYM
  // check results
  unsigned idx = 0;
  for (unsigned i=0; i<N; ++i) {
    if (PREDICATE(in[i])) {
      assert(out[idx] == in[i]);
      idx++;
    }
  }
  printf("TEST PASSED\n");
#endif

  // cleanup
  free(in);
  free(out);
  hipFree(d_in);
  hipFree(d_out);
  return 0;
}
