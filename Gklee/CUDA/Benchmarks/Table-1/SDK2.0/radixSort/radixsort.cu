#include "hip/hip_runtime.h"
/*
* Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and 
* proprietary rights in and to this software and related documentation and 
* any modifications thereto.  Any use, reproduction, disclosure, or distribution 
* of this software and related documentation without an express license 
* agreement from NVIDIA Corporation is strictly prohibited.
* 
*/

// -----------------------------------------------------------------------
// Fast CUDA Radix Sort Implementation
//
// The parallel radix sort algorithm implemented by this code is described
// in the following paper.
//
// Satish, N., Harris, M., and Garland, M. "Designing Efficient Sorting 
// Algorithms for Manycore GPUs". In Proceedings of IEEE International
// Parallel & Distributed Processing Symposium 2009 (IPDPS 2009).
//
// -----------------------------------------------------------------------

#include "radixsort.h"

#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <assert.h>

#if (CUDART_VERSION  < 2020)
#error CUDA runtime version 2.2 or later required!
#endif

// Used for creating a mapping of kernel functions to the number of CTAs to launch for each
typedef void* KernelPointer;
extern "C" int getNumCTAs(KernelPointer kernel);
extern "C" void setNumCTAs(KernelPointer kernel, int numCTAs);
extern "C" void computeNumCTAs(KernelPointer kernel, int smemDynamicBytes, bool bManualCoalesce);

bool bManualCoalesce = false;
unsigned int persistentCTAThreshold[2] = { 0, 0 };
unsigned int persistentCTAThresholdFullBlocks[2] = { 0, 0 };

template <typename T>
int numCTAs(T kernel)
{
    return getNumCTAs((KernelPointer)kernel);
}

template <typename T>
void numCTAs(T kernel, int numCTAs)
{
    setNumCTAs((KernelPointer)kernel, numCTAs);
}

template <typename T>
void computeNumCTAs(T kernel, int smemDynamicBytes)
{
    computeNumCTAs((KernelPointer)kernel, smemDynamicBytes, bManualCoalesce);
}

// In emulation mode, we need __syncthreads() inside warp-synchronous code,
// but we don't in code running on the GPU, so we define this macro to use
// in the warp-scan portion of the radix sort (see CUDPP for information
// on the warp scan algorithm.
#ifdef __DEVICE_EMULATION__
#define __SYNC __syncthreads();
#else
#define __SYNC
#endif

typedef unsigned int uint;

extern "C"
void checkCudaError(const char *msg)
{
#if defined(_DEBUG) || defined(DEBUG)
    hipError_t e = hipDeviceSynchronize();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error %s : %s\n", msg, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
    e = hipGetLastError();
    if( e != hipSuccess )
    {
        fprintf(stderr, "CUDA Error %s : %s\n", msg, hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }
#endif
}

__global__ void emptyKernel() {}

// -----------------------------------------------------------------------------------------------
// The floatFlip and floatUnflip functions below are based on code in the web article 
// "Radix Tricks" by Michael Herf (http://www.stereopsis.com/radix.html). They are used to convert
// floating point values into sortable unsigned integers (and back).
//
// Paraphrasing Michael: Binary single-precision floating point numbers have two features that 
// keep them from being directly sortable. First, the sign bit is set when the value is negative, 
// which means that all negative numbers are bigger than positive ones. Second, the values are 
// signed-magnitude, so "more negative" floating point numbers actually look bigger to a normal 
// bitwise comparison.
// 
// "To fix our floating point numbers, we define the following rules:
//
//   1. Always flip the sign bit.
//   2. If the sign bit was set, flip the other bits too.
//
// To get back, we flip the sign bit always, and if the sign bit was not set, we flip the other 
// bits too."
//
// This is a very inexpensive operation and it is only done on the first and last steps of the
// sort.
// -----------------------------------------------------------------------------------------------


// ================================================================================================
// Flip a float for sorting
//  finds SIGN of fp number.
//  if it's 1 (negative float), it flips all bits
//  if it's 0 (positive float), it flips the sign only
// ================================================================================================
template <bool doFlip>
__device__ uint floatFlip(uint f)
{
    if (doFlip)
    {
        uint mask = -int(f >> 31) | 0x80000000;
	return f ^ mask;
    }
    else
        return f;
}

// ================================================================================================
// flip a float back (invert FloatFlip)
//  signed was flipped from above, so:
//  if sign is 1 (negative), it flips the sign bit back
//  if sign is 0 (positive), it flips all bits back
// ================================================================================================
template <bool doFlip>
__device__ uint floatUnflip(uint f)
{
    if (doFlip)
    {
        uint mask = ((f >> 31) - 1) | 0x80000000;
	    return f ^ mask;
    }
    else
        return f;
}

// ================================================================================================
// Kernel to flip all floats in an array (see floatFlip, above)
// Each thread flips four values (each 256-thread CTA flips 1024 values).
// ================================================================================================
__global__ void flipFloats(uint *values, uint numValues)
{
    uint index = __umul24(blockDim.x*4, blockIdx.x) + threadIdx.x; 
    if (index < numValues) values[index] = floatFlip<true>(values[index]);
    index += blockDim.x;
    if (index < numValues) values[index] = floatFlip<true>(values[index]);
    index += blockDim.x;
    if (index < numValues) values[index] = floatFlip<true>(values[index]);
    index += blockDim.x;
    if (index < numValues) values[index] = floatFlip<true>(values[index]);
}

// ================================================================================================
// Kernel to unflip all floats in an array (see floatUnflip, above)
// Each thread unflips four values (each 256-thread CTA unflips 1024 values).
// ================================================================================================
__global__ void unflipFloats(uint *values, uint numValues)
{
    uint index = __umul24(blockDim.x*4, blockIdx.x) + threadIdx.x; 
    if (index < numValues) values[index] = floatUnflip<true>(values[index]);
    index += blockDim.x;
    if (index < numValues) values[index] = floatUnflip<true>(values[index]);
    index += blockDim.x;
    if (index < numValues) values[index] = floatUnflip<true>(values[index]);
    index += blockDim.x;
    if (index < numValues) values[index] = floatUnflip<true>(values[index]);
}

//----------------------------------------------------------------------------
// Scans each warp in parallel ("warp-scan"), one element per thread.
// uses 2 numElements of shared memory per thread (64 = elements per warp)
//----------------------------------------------------------------------------
template<class T, int maxlevel>
__device__ T scanwarp(T val, T* sData)
{
    // The following is the same as 2 * RadixSort::WARP_SIZE * warpId + threadInWarp = 
    // 64*(threadIdx.x >> 5) + (threadIdx.x & (RadixSort::WARP_SIZE - 1))
    int idx = 2 * threadIdx.x - (threadIdx.x & (RadixSort::WARP_SIZE - 1));
    sData[idx] = 0;
    idx += RadixSort::WARP_SIZE;
    sData[idx] = val;          __SYNC

#ifdef __DEVICE_EMULATION__
        T t = sData[idx -  1]; __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx -  2];   __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx -  4];   __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx -  8];   __SYNC 
        sData[idx] += t;       __SYNC
        t = sData[idx - 16];   __SYNC 
        sData[idx] += t;       __SYNC
#else
        if (0 <= maxlevel) { sData[idx] += sData[idx - 1]; } __SYNC
        if (1 <= maxlevel) { sData[idx] += sData[idx - 2]; } __SYNC
        if (2 <= maxlevel) { sData[idx] += sData[idx - 4]; } __SYNC
        if (3 <= maxlevel) { sData[idx] += sData[idx - 8]; } __SYNC
        if (4 <= maxlevel) { sData[idx] += sData[idx -16]; } __SYNC
#endif

        return sData[idx] - val;  // convert inclusive -> exclusive
}

//----------------------------------------------------------------------------
// scan4 scans 4*RadixSort::CTA_SIZE numElements in a block (4 per thread), using 
// a warp-scan algorithm
//----------------------------------------------------------------------------
__device__ uint4 scan4(uint4 idata)
{    
    extern uint ptr[];
    uint idx = threadIdx.x;

    uint4 val4 = idata;
    uint sum[3];
    sum[0] = val4.x;
    sum[1] = val4.y + sum[0];
    sum[2] = val4.z + sum[1];
    
    uint val = val4.w + sum[2];
    
    val = scanwarp<uint, 4>(val, ptr);
    __syncthreads();

    if ((idx & (RadixSort::WARP_SIZE - 1)) == RadixSort::WARP_SIZE - 1)
    {
        ptr[idx >> 5] = val + val4.w + sum[2];
    }
    __syncthreads();

#ifndef __DEVICE_EMULATION__
    if (idx < RadixSort::WARP_SIZE)
#endif
    {
        ptr[idx] = scanwarp<uint, 2>(ptr[idx], ptr);
    }
    __syncthreads();

    val += ptr[idx >> 5];

    val4.x = val;
    val4.y = val + sum[0];
    val4.z = val + sum[1];
    val4.w = val + sum[2];

    return val4;
}

//----------------------------------------------------------------------------
//
// Rank is the core of the radix sort loop.  Given a predicate, it
// computes the output position for each thread in an ordering where all
// True threads come first, followed by all False threads.
// 
// This version handles 4 predicates per thread; hence, "rank4".
//
//----------------------------------------------------------------------------
template <int ctasize>
__device__ uint4 rank4(uint4 preds)
{
    uint4 address = scan4(preds);  

    __shared__ uint numtrue;
    if (threadIdx.x == ctasize-1)
    {
        numtrue = address.w + preds.w;
    }
    __syncthreads();

    uint4 rank;
    uint idx = threadIdx.x << 2;
    rank.x = (preds.x) ? address.x : numtrue + idx   - address.x;
    rank.y = (preds.y) ? address.y : numtrue + idx + 1 - address.y;
    rank.z = (preds.z) ? address.z : numtrue + idx + 2 - address.z;
    rank.w = (preds.w) ? address.w : numtrue + idx + 3 - address.w;	

    return rank;
}

//----------------------------------------------------------------------------
// Uses rank to sort one bit at a time: Sorts a block according
// to bits startbit -> nbits + startbit
//
// Each thread sorts 4 elements by nbits bits
//----------------------------------------------------------------------------
    __extern__shared__ uint sMem1[];
template<uint nbits, uint startbit>
__device__ void radixSortBlock(uint4 &key, uint4 &value)
{

    for(uint shift = startbit; shift < (startbit + nbits); ++shift)
    {        
        uint4 lsb;
        lsb.x = !((key.x >> shift) & 0x1);
        lsb.y = !((key.y >> shift) & 0x1);
        lsb.z = !((key.z >> shift) & 0x1);
        lsb.w = !((key.w >> shift) & 0x1);

        uint4 r = rank4<RadixSort::CTA_SIZE>(lsb);

        // This arithmetic strides the ranks across 4 CTA_SIZE regions
        sMem1[(r.x & 3) * RadixSort::CTA_SIZE + (r.x >> 2)] = key.x;
        sMem1[(r.y & 3) * RadixSort::CTA_SIZE + (r.y >> 2)] = key.y;
        sMem1[(r.z & 3) * RadixSort::CTA_SIZE + (r.z >> 2)] = key.z;
        sMem1[(r.w & 3) * RadixSort::CTA_SIZE + (r.w >> 2)] = key.w;
        __syncthreads();

        // The above allows us to read without 4-way bank conflicts:
        key.x = sMem1[threadIdx.x];
        key.y = sMem1[threadIdx.x +     RadixSort::CTA_SIZE];
        key.z = sMem1[threadIdx.x + 2 * RadixSort::CTA_SIZE];
        key.w = sMem1[threadIdx.x + 3 * RadixSort::CTA_SIZE];

        __syncthreads();

        sMem1[(r.x & 3) * RadixSort::CTA_SIZE + (r.x >> 2)] = value.x;
        sMem1[(r.y & 3) * RadixSort::CTA_SIZE + (r.y >> 2)] = value.y;
        sMem1[(r.z & 3) * RadixSort::CTA_SIZE + (r.z >> 2)] = value.z;
        sMem1[(r.w & 3) * RadixSort::CTA_SIZE + (r.w >> 2)] = value.w;
        __syncthreads();

        value.x = sMem1[threadIdx.x];
        value.y = sMem1[threadIdx.x +     RadixSort::CTA_SIZE];
        value.z = sMem1[threadIdx.x + 2 * RadixSort::CTA_SIZE];
        value.w = sMem1[threadIdx.x + 3 * RadixSort::CTA_SIZE];

        __syncthreads();
    }
}

//----------------------------------------------------------------------------
//
// radixSortBlocks sorts all blocks of data independently in shared
// memory.  Each thread block (CTA) sorts one block of 4*CTA_SIZE elements
//
// The radix sort is done in two stages.  This stage calls radixSortBlock on each 
// block independently, sorting on the basis of bits (startbit) -> (startbit + nbits)
// 
// Template parameters are used to generate efficient code for various special cases
// For example, we have to handle arrays that are a multiple of the block size (fullBlocks)
// differently than arrays that are not.  "flip" is used to only compile in the
// float flip code when float keys are used.  "loop" is used when persistent CTAs
// are used. 
//
// By persistent CTAs we mean that we launch only as many thread blocks as can 
// be resident in the GPU and no more, rather than launching as many threads as
// we have elements. Persistent CTAs loop over blocks of elements until all work
// is complete.  This can be faster in some cases.  In our tests it is faster
// for large sorts (and the threshold is higher on compute version 1.1 and earlier
// GPUs than it is on compute version 1.2 GPUs.
//----------------------------------------------------------------------------
__extern__shared__ uint4 sMem[];
template<uint nbits, uint startbit, bool fullBlocks, bool flip, bool loop>
__global__ void radixSortBlocks(uint4* keysOut, uint4* valuesOut, 
                                uint4* keysIn, uint4* valuesIn, 
                                uint numElements, uint totalBlocks)
{

    uint4 key, value;

    
    uint blockId = blockIdx.x;

    while (!loop || blockId < totalBlocks)
    {
        uint i = blockId * blockDim.x + threadIdx.x;
        uint idx = i << 2;

        // handle non-full last block if array is not multiple of 1024 numElements
        if (!fullBlocks && idx+3 >= numElements)
        {
            if (idx >= numElements)
            {
                key   = make_uint4(UINT_MAX, UINT_MAX, UINT_MAX, UINT_MAX);
                value = make_uint4(UINT_MAX, UINT_MAX, UINT_MAX, UINT_MAX);
            }
            else
            {
                // for non-full block, we handle uint1 values instead of uint4
                uint *keys1    = (uint*)keysIn;
                uint *values1  = (uint*)valuesIn;

                key.x = (idx   < numElements) ? floatFlip<flip>(keys1[idx])   : UINT_MAX;
                key.y = (idx+1 < numElements) ? floatFlip<flip>(keys1[idx+1]) : UINT_MAX;
                key.z = (idx+2 < numElements) ? floatFlip<flip>(keys1[idx+2]) : UINT_MAX;
                key.w = UINT_MAX;

                value.x = (idx   < numElements) ? values1[idx]   : UINT_MAX;
                value.y = (idx+1 < numElements) ? values1[idx+1] : UINT_MAX;
                value.z = (idx+2 < numElements) ? values1[idx+2] : UINT_MAX;
                value.w = UINT_MAX;
            }
        }
        else
        {
            key = keysIn[i];
            value = valuesIn[i];

            if (flip)
            {
                key.x = floatFlip<flip>(key.x);
                key.y = floatFlip<flip>(key.y);
                key.z = floatFlip<flip>(key.z);
                key.w = floatFlip<flip>(key.w);
            }
        }
        __syncthreads();
        radixSortBlock<nbits, startbit>(key, value);

        // handle non-full last block if array is not multiple of 1024 numElements
        if(!fullBlocks && idx+3 >= numElements)
        {
            if (idx < numElements) 
            {
                // for non-full block, we handle uint1 values instead of uint4
                uint *keys1   = (uint*)keysOut;
                uint *values1 = (uint*)valuesOut;

                keys1[idx]   = key.x;
                values1[idx] = value.x;

                if (idx + 1 < numElements)
                {
                    keys1[idx + 1]   = key.y;
                    values1[idx + 1] = value.y;

                    if (idx + 2 < numElements)
                    {
                        keys1[idx + 2]   = key.z;
                        values1[idx + 2] = value.z;
                    }
                }
            }
        }
        else
        {
            keysOut[i]   = key;
            valuesOut[i] = value;
        }
        
        if (loop)        
            blockId += gridDim.x;
        else
            break;            
    }
}

//----------------------------------------------------------------------------
// Given an array with blocks sorted according to a 4-bit radix group, each 
// block counts the number of keys that fall into each radix in the group, and 
// finds the starting offset of each radix in the block.  It then writes the radix 
// counts to the counters array, and the starting offsets to the blockOffsets array.
//
// Template parameters are used to generate efficient code for various special cases
// For example, we have to handle arrays that are a multiple of the block size 
// (fullBlocks) differently than arrays that are not. "loop" is used when persistent 
// CTAs are used. 
//
// By persistent CTAs we mean that we launch only as many thread blocks as can 
// be resident in the GPU and no more, rather than launching as many threads as
// we have elements. Persistent CTAs loop over blocks of elements until all work
// is complete.  This can be faster in some cases.  In our tests it is faster
// for large sorts (and the threshold is higher on compute version 1.1 and earlier
// GPUs than it is on compute version 1.2 GPUs.
//                                
//----------------------------------------------------------------------------
    __extern__shared__ uint sRadix1[];
template<uint startbit, bool fullBlocks, bool loop>
__global__ void findRadixOffsets(uint2 *keys, 
                                 uint  *counters, 
                                 uint  *blockOffsets, 
                                 uint   numElements,
                                 uint   totalBlocks)
{
    __shared__ uint  sStartPointers[16];

    uint blockId = blockIdx.x;   

    while (!loop || blockId < totalBlocks)
    {
        uint2 radix2;

        uint i       = blockId * blockDim.x + threadIdx.x;

        // handle non-full last block if array is not multiple of 1024 numElements
        if(!fullBlocks && ((i + 1) << 1 ) > numElements )
        {
            // handle uint1 rather than uint2 for non-full blocks
            uint *keys1 = (uint*)keys;
            uint j = i << 1; 

            radix2.x = (j < numElements) ? keys1[j] : UINT_MAX; 
            j++;
            radix2.y = (j < numElements) ? keys1[j] : UINT_MAX;
        }
        else
        {
            radix2 = keys[i];
        }

        sRadix1[2 * threadIdx.x]     = (radix2.x >> startbit) & 0xF;
        sRadix1[2 * threadIdx.x + 1] = (radix2.y >> startbit) & 0xF;

        // Finds the position where the sRadix1 entries differ and stores start 
        // index for each radix.
        if(threadIdx.x < 16) 
        { 
            sStartPointers[threadIdx.x] = 0; 
        }
        __syncthreads();

        if((threadIdx.x > 0) && (sRadix1[threadIdx.x] != sRadix1[threadIdx.x - 1]) ) 
        {
            sStartPointers[sRadix1[threadIdx.x]] = threadIdx.x;
        }
        if(sRadix1[threadIdx.x + RadixSort::CTA_SIZE] != sRadix1[threadIdx.x + RadixSort::CTA_SIZE - 1]) 
        {
            sStartPointers[sRadix1[threadIdx.x + RadixSort::CTA_SIZE]] = threadIdx.x + RadixSort::CTA_SIZE;
        }
        __syncthreads();

        if(threadIdx.x < 16) 
        {
            blockOffsets[blockId*16 + threadIdx.x] = sStartPointers[threadIdx.x];
        }
        __syncthreads();

        // Compute the sizes of each block.
        if((threadIdx.x > 0) && (sRadix1[threadIdx.x] != sRadix1[threadIdx.x - 1]) ) 
        {
            sStartPointers[sRadix1[threadIdx.x - 1]] = 
                threadIdx.x - sStartPointers[sRadix1[threadIdx.x - 1]];
        }
        if(sRadix1[threadIdx.x + RadixSort::CTA_SIZE] != sRadix1[threadIdx.x + RadixSort::CTA_SIZE - 1] ) 
        {
            sStartPointers[sRadix1[threadIdx.x + RadixSort::CTA_SIZE - 1]] = 
                threadIdx.x + RadixSort::CTA_SIZE - sStartPointers[sRadix1[threadIdx.x + RadixSort::CTA_SIZE - 1]];
        }
        

        if(threadIdx.x == RadixSort::CTA_SIZE - 1) 
        {
            sStartPointers[sRadix1[2 * RadixSort::CTA_SIZE - 1]] = 
                2 * RadixSort::CTA_SIZE - sStartPointers[sRadix1[2 * RadixSort::CTA_SIZE - 1]];
        }
        __syncthreads();

        if(threadIdx.x < 16) 
        {
            counters[threadIdx.x * totalBlocks + blockId] = 
                sStartPointers[threadIdx.x];
        }

        if (loop)
            blockId += gridDim.x;
        else
            break;
    }
}

//----------------------------------------------------------------------------
// reorderData shuffles data in the array globally after the radix offsets 
// have been found. On compute version 1.1 and earlier GPUs, this code depends 
// on RadixSort::CTA_SIZE being 16 * number of radices (i.e. 16 * 2^nbits).
// 
// On compute version 1.1 GPUs ("manualCoalesce=true") this function ensures
// that all writes are coalesced using extra work in the kernel.  On later
// GPUs coalescing rules have been relaxed, so this extra overhead hurts 
// performance.  On these GPUs we set manualCoalesce=false and directly store
// the results.
//
// Template parameters are used to generate efficient code for various special cases
// For example, we have to handle arrays that are a multiple of the block size 
// (fullBlocks) differently than arrays that are not.  "loop" is used when persistent 
// CTAs are used. 
//
// By persistent CTAs we mean that we launch only as many thread blocks as can 
// be resident in the GPU and no more, rather than launching as many threads as
// we have elements. Persistent CTAs loop over blocks of elements until all work
// is complete.  This can be faster in some cases.  In our tests it is faster
// for large sorts (and the threshold is higher on compute version 1.1 and earlier
// GPUs than it is on compute version 1.2 GPUs.
//----------------------------------------------------------------------------
template<uint startbit, bool fullBlocks, bool manualCoalesce, bool unflip, bool loop>
__global__ void reorderData(uint  *outKeys, 
                            uint  *outValues, 
                            uint2 *keys, 
                            uint2 *values, 
                            uint  *blockOffsets, 
                            uint  *offsets, 
                            uint  *sizes, 
                            uint   numElements,
                            uint   totalBlocks)
{
    __shared__ uint2 sKeys2[RadixSort::CTA_SIZE];
    __shared__ uint2 sValues2[RadixSort::CTA_SIZE];
    __shared__ uint sOffsets[16];
    __shared__ uint sBlockOffsets[16];

    uint *sKeys1   = (uint*)sKeys2; 
    uint *sValues1 = (uint*)sValues2; 

    uint blockId = blockIdx.x;   

    while (!loop || blockId < totalBlocks)
    {
        uint i = blockId * blockDim.x + threadIdx.x;

        // handle non-full last block if array is not multiple of 1024 numElements
        if(!fullBlocks && (((i + 1) << 1) > numElements))
        {
            uint *keys1   = (uint*)keys;
            uint *values1 = (uint*)values;
            uint j = i << 1; 

            sKeys1[threadIdx.x << 1]   = (j < numElements) ? keys1[j]   : UINT_MAX; 
            sValues1[threadIdx.x << 1] = (j < numElements) ? values1[j] : UINT_MAX; 
            j++; 
            sKeys1[(threadIdx.x << 1) + 1]   = (j < numElements) ? keys1[j]   : UINT_MAX; 
            sValues1[(threadIdx.x << 1) + 1] = (j < numElements) ? values1[j] : UINT_MAX; 
        }
        else
        {
            sKeys2[threadIdx.x]   = keys[i];
            sValues2[threadIdx.x] = values[i];
        }

        if (!manualCoalesce)
        {
            if(threadIdx.x < 16)  
            {
                sOffsets[threadIdx.x]      = offsets[threadIdx.x * totalBlocks + blockId];
                sBlockOffsets[threadIdx.x] = blockOffsets[blockId * 16 + threadIdx.x];
            }
            __syncthreads();

            uint radix = (sKeys1[threadIdx.x] >> startbit) & 0xF;
            uint globalOffset = sOffsets[radix] + threadIdx.x - sBlockOffsets[radix];

            if (fullBlocks || globalOffset < numElements)
            {
                outKeys[globalOffset]   = floatUnflip<unflip>(sKeys1[threadIdx.x]);
                outValues[globalOffset] = sValues1[threadIdx.x];
            }

            radix = (sKeys1[threadIdx.x + RadixSort::CTA_SIZE] >> startbit) & 0xF;
            globalOffset = sOffsets[radix] + threadIdx.x + RadixSort::CTA_SIZE - sBlockOffsets[radix];

            if (fullBlocks || globalOffset < numElements)
            {
                outKeys[globalOffset]   = floatUnflip<unflip>(sKeys1[threadIdx.x + RadixSort::CTA_SIZE]);
                outValues[globalOffset] = sValues1[threadIdx.x + RadixSort::CTA_SIZE];
            }
        }
        else
        {
            __shared__ uint sSizes[16];

            if(threadIdx.x < 16)  
            {
                sOffsets[threadIdx.x]      = offsets[threadIdx.x * totalBlocks + blockId];
                sBlockOffsets[threadIdx.x] = blockOffsets[blockId * 16 + threadIdx.x];
                sSizes[threadIdx.x]        = sizes[threadIdx.x * totalBlocks + blockId];
            }
            __syncthreads();

            // 1 half-warp is responsible for writing out all values for 1 radix. 
            // Loops if there are more than 16 values to be written out. 
            // All start indices are rounded down to the nearest multiple of 16, and
            // all end indices are rounded up to the nearest multiple of 16.
            // Thus it can do extra work if the start and end indices are not multiples of 16
            // This is bounded by a factor of 2 (it can do 2X more work at most).

            const uint halfWarpID     = threadIdx.x >> 4;

            const uint halfWarpOffset = threadIdx.x & 0xF;
            const uint leadingInvalid = sOffsets[halfWarpID] & 0xF;

            uint startPos = sOffsets[halfWarpID] & 0xFFFFFFF0;
            uint endPos   = (sOffsets[halfWarpID] + sSizes[halfWarpID]) + 15 - 
                ((sOffsets[halfWarpID] + sSizes[halfWarpID] - 1) & 0xF);
            uint numIterations = endPos - startPos;

            uint outOffset = startPos + halfWarpOffset;
            uint inOffset  = sBlockOffsets[halfWarpID] - leadingInvalid + halfWarpOffset;

            for(uint j = 0; j < numIterations; j += 16, outOffset += 16, inOffset += 16)
            {       
                if( (outOffset >= sOffsets[halfWarpID]) && 
                    (inOffset - sBlockOffsets[halfWarpID] < sSizes[halfWarpID])) 
                {
                    if(blockId < totalBlocks - 1 || outOffset < numElements) 
                    {
                        outKeys[outOffset]   = floatUnflip<unflip>(sKeys1[inOffset]);
                        outValues[outOffset] = sValues1[inOffset];
                    }
                }       
            }
        }

        if (loop)
        {
            blockId += gridDim.x;
            __syncthreads();
        }
        else
            break;
    }
}

//----------------------------------------------------------------------------
// Perform one step of the radix sort.  Sorts by nbits key bits per step, 
// starting at startbit.
//
// Uses cudppScan() for the prefix sum of radix counters.
//----------------------------------------------------------------------------
template<uint nbits, uint startbit, bool flip, bool unflip>
void radixSortStep(uint *keys, 
                   uint *values, 
                   uint *tempKeys, 
                   uint *tempValues, 
                   uint *counters, 
                   uint *countersSum, 
                   uint *blockOffsets, 
                   CUDPPHandle scanPlan,
                   uint numElements)
{
    const uint eltsPerBlock = RadixSort::CTA_SIZE * 4;
    const uint eltsPerBlock2 = RadixSort::CTA_SIZE * 2;

    bool fullBlocks = ((numElements % eltsPerBlock) == 0);
    uint numBlocks = (fullBlocks) ? 
        (numElements / eltsPerBlock) : 
        (numElements / eltsPerBlock + 1);
    uint numBlocks2 = ((numElements % eltsPerBlock2) == 0) ?
        (numElements / eltsPerBlock2) : 
        (numElements / eltsPerBlock2 + 1);
  
    bool loop = numBlocks > 65535;
    //bool loop2 = numBlocks2 > 65535;
    uint blocks = loop ? 65535 : numBlocks;
    uint blocksFind = loop ? 65535 : numBlocks2;
    uint blocksReorder = loop ? 65535 : numBlocks2;

    uint threshold = fullBlocks ? persistentCTAThresholdFullBlocks[0] : persistentCTAThreshold[0];

    if (numElements >= threshold)
    {
        loop = (numElements > 262144) || (numElements >= 32768 && numElements < 65536);
        
        blocks = numBlocks;
        blocksFind = numBlocks2;
        blocksReorder = numBlocks2;

        // Run an empty kernel -- this seems to reset some of the CTA scheduling hardware
	    // on GT200, resulting in better scheduling and lower run times
        if (startbit > 0)
        {
            emptyKernel<<<numCTAs(emptyKernel), RadixSort::CTA_SIZE>>>();
        }
    }
    
    if (fullBlocks)
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocks = flip? numCTAs(radixSortBlocks<4, 0, true, true, true>) : numCTAs(radixSortBlocks<4, 0, true, false, true>);
            }

            radixSortBlocks<nbits, startbit, true, flip, true>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)tempValues, (uint4*)keys, (uint4*)values, numElements, numBlocks);
        }
        else
        {
            radixSortBlocks<nbits, startbit, true, flip, false>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)tempValues, (uint4*)keys, (uint4*)values, numElements, numBlocks);
        }
    }
    else
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocks = flip ? numCTAs(radixSortBlocks<4, 0, false, true, true>) : numCTAs(radixSortBlocks<4, 0, false, false, true>);
            }

            radixSortBlocks<nbits, startbit, false, flip, true>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)tempValues, (uint4*)keys, (uint4*)values, numElements, numBlocks);
        }
        else
        {
            radixSortBlocks<nbits, startbit, false, flip, false>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)tempValues, (uint4*)keys, (uint4*)values, numElements, numBlocks);
        }
    }

    checkCudaError("radixSortBlocks");

    if (fullBlocks)
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocksFind = numCTAs(findRadixOffsets<0, true, true>);
            }
            findRadixOffsets<startbit, true, true>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
        }
        else
        {
            findRadixOffsets<startbit, true, false>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
        }
    }
    else
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocksFind = numCTAs(findRadixOffsets<0, false, true>);
            }
            findRadixOffsets<startbit, false, true>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
        }
        else
        {
            findRadixOffsets<startbit, false, false>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
        }
    }

    checkCudaError("findRadixOffsets");

    //cudppScan(scanPlan, countersSum, counters, 16*numBlocks2);

    if (fullBlocks)
    {
        if (bManualCoalesce)
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ? numCTAs(reorderData<0, true, true, true, true>) :
                        	    	     numCTAs(reorderData<0, true, true, false, true>);
                }
                reorderData<startbit, true, true, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
            else
            {
                reorderData<startbit, true, true, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
        }
        else
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ? numCTAs(reorderData<0, true, false, true, true>) :
		    		             numCTAs(reorderData<0, true, false, false, true>);
                }
                reorderData<startbit, true, false, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
            else
            {
                reorderData<startbit, true, false, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
        }
    }
    else
    {
        if (bManualCoalesce)
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ? 
                        numCTAs(reorderData<0, false, true, true, true>) :
                        numCTAs(reorderData<0, false, true, false, true>);
                }
                reorderData<startbit, false, true, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
            else
            {
                reorderData<startbit, false, true, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
        }
        else
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ?
                        numCTAs(reorderData<0, false, false, true, true>) :
                        numCTAs(reorderData<0, false, false, false, true>);
                }
                reorderData<startbit, false, false, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
            else
            {
                reorderData<startbit, false, false, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, values, (uint2*)tempKeys, (uint2*)tempValues, 
                    blockOffsets, countersSum, counters, numElements, numBlocks2);
            }
        }
    }

    checkCudaError("radixSortStep");
}

//----------------------------------------------------------------------------
// Optimization for sorts of fewer than 4 * CTA_SIZE elements
//----------------------------------------------------------------------------
template <bool flip>
void radixSortSingleBlock(uint *keys, 
                          uint *values, 
                          uint numElements)
{
    bool fullBlocks = (numElements % (RadixSort::CTA_SIZE * 4) == 0);
    if (fullBlocks)
    {
        radixSortBlocks<32, 0, true, flip, false>
            <<<1, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)keys, (uint4*)values, 
                 (uint4*)keys, (uint4*)values, 
                 numElements, 1 );
    }
    else
    {
        radixSortBlocks<32, 0, false, flip, false>
            <<<1, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)keys, (uint4*)values, 
                 (uint4*)keys, (uint4*)values, 
                 numElements, 1 );
    }

    if (flip)
            unflipFloats<<<1, RadixSort::CTA_SIZE>>>(keys, numElements);

    checkCudaError("radixSortSingleBlock");
}

//----------------------------------------------------------------------------
// Optimization for sorts of WARP_SIZE or fewer elements
//----------------------------------------------------------------------------
template <bool flip>
__global__ 
void radixSortSingleWarp(uint *keys, 
                         uint *values, 
                         uint numElements)
{
    volatile __shared__ uint sKeys[RadixSort::WARP_SIZE];
    volatile __shared__ uint sValues[RadixSort::WARP_SIZE];
    volatile __shared__ uint sFlags[RadixSort::WARP_SIZE];

    sKeys[threadIdx.x]   = floatFlip<flip>(keys[threadIdx.x]);
    sValues[threadIdx.x] = values[threadIdx.x];
    
    __SYNC // emulation only

    for(uint i = 1; i < numElements; i++)
    {
        uint key_i = sKeys[i];
        uint val_i = sValues[i];
        
        sFlags[threadIdx.x] = 0;
        
        if( (threadIdx.x < i) && (sKeys[threadIdx.x] > key_i) ) 
        {
            uint temp = sKeys[threadIdx.x];
            uint tempval = sValues[threadIdx.x];
            sFlags[threadIdx.x] = 1;
            sKeys[threadIdx.x + 1] = temp;
            sValues[threadIdx.x + 1] = tempval;
            sFlags[threadIdx.x + 1] = 0;
        }
        if(sFlags[threadIdx.x] == 1 )
        {
            sKeys[threadIdx.x] = key_i;
            sValues[threadIdx.x] = val_i;
        }

        __SYNC // emulation only

    }
    keys[threadIdx.x]   = floatUnflip<flip>(sKeys[threadIdx.x]);
    values[threadIdx.x] = sValues[threadIdx.x];
}

//----------------------------------------------------------------------------
// Main radix sort function.  Sorts in place in the keys and values arrays,
// but uses the other device arrays as temporary storage.  All pointer 
// parameters are device pointers.  
//----------------------------------------------------------------------------
void radixSort(uint *keys, 
               uint *values, 
               uint *tempKeys, 
               uint *tempValues,
               uint *counters,
               uint *countersSum,
               uint *blockOffsets,
               CUDPPHandle scanPlan,
               uint numElements, 
               uint keyBits,
               bool flipBits = false)
{
    if(numElements <= RadixSort::WARP_SIZE)
    {
        if (flipBits)
            radixSortSingleWarp<true><<<1, numElements>>>(keys, values, numElements);
        else
            radixSortSingleWarp<false><<<1, numElements>>>(keys, values, numElements);
        checkCudaError("radixSortSingleWarp");
        return;
    }
    if(numElements <= RadixSort::CTA_SIZE * 4)
    {
        if (flipBits)
            radixSortSingleBlock<true>(keys, values, numElements);
        else
            radixSortSingleBlock<false>(keys, values, numElements);
        return;
    }

    // flip float bits on the first pass, unflip on the last pass
    if (flipBits) 
    {
            radixSortStep<4,  0, true, false>(keys, values, tempKeys, tempValues, 
                                              counters, countersSum, blockOffsets, 
                                              scanPlan, numElements);
    }
    else
    {       radixSortStep<4,  0, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }

    if (keyBits > 4)
    {
            radixSortStep<4,  4, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }
    if (keyBits > 8)
    {
            radixSortStep<4,  8, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }
    if (keyBits > 12)
    {
            radixSortStep<4, 12, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }
    if (keyBits > 16)
    {
            radixSortStep<4, 16, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }
    if (keyBits > 20)
    {
            radixSortStep<4, 20, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }
    if (keyBits > 24)
    {
            radixSortStep<4, 24, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
    }
    if (keyBits > 28)
    {
        if (flipBits) // last pass
        {
            radixSortStep<4, 28, false, true>(keys, values, tempKeys, tempValues, 
                                              counters, countersSum, blockOffsets, 
                                              scanPlan, numElements);
        }
        else
        {
            radixSortStep<4, 28, false, false>(keys, values, tempKeys, tempValues, 
                                               counters, countersSum, blockOffsets, 
                                               scanPlan, numElements);
        }
    }

    checkCudaError("radixSort");
}

extern "C"
void radixSortFloatKeys(float *keys, 
                        uint  *values, 
                        float *tempKeys, 
                        uint  *tempValues,
                        uint  *counters,
                        uint  *countersSum,
                        uint  *blockOffsets,
                        CUDPPHandle scanPlan,
                        uint  numElements, 
                        uint  keyBits,
                        bool  negativeKeys)
{
    radixSort((uint*)keys, values, (uint*)tempKeys, tempValues, counters, 
              countersSum, blockOffsets, scanPlan, numElements, keyBits, 
              negativeKeys);
    checkCudaError("radixSortFloatKeys");
}

//----------------------------------------------------------------------------
// Key-only Sorts
//----------------------------------------------------------------------------

//----------------------------------------------------------------------------
// Uses rank to sort one bit at a time: Sorts a block according
// to bits startbit -> nbits + startbit
//----------------------------------------------------------------------------
    __extern__shared__ uint sMem1[];
template<uint nbits, uint startbit>
__device__ void radixSortBlockKeysOnly(uint4 &key)
{

    for(uint shift = startbit; shift < (startbit + nbits); ++shift)
    {        
        uint4 lsb;
        lsb.x = !((key.x >> shift) & 0x1);
        lsb.y = !((key.y >> shift) & 0x1);
        lsb.z = !((key.z >> shift) & 0x1);
        lsb.w = !((key.w >> shift) & 0x1);

        uint4 r = rank4<256>(lsb);

#if 1
        // This arithmetic strides the ranks across 4 CTA_SIZE regions
        sMem1[(r.x & 3) * RadixSort::CTA_SIZE + (r.x >> 2)] = key.x;
        sMem1[(r.y & 3) * RadixSort::CTA_SIZE + (r.y >> 2)] = key.y;
        sMem1[(r.z & 3) * RadixSort::CTA_SIZE + (r.z >> 2)] = key.z;
        sMem1[(r.w & 3) * RadixSort::CTA_SIZE + (r.w >> 2)] = key.w;
        __syncthreads();

        // The above allows us to read without 4-way bank conflicts:
        key.x = sMem1[threadIdx.x];
        key.y = sMem1[threadIdx.x +     RadixSort::CTA_SIZE];
        key.z = sMem1[threadIdx.x + 2 * RadixSort::CTA_SIZE];
        key.w = sMem1[threadIdx.x + 3 * RadixSort::CTA_SIZE];
#else
        sMem1[r.x] = key.x;
        sMem1[r.y] = key.y;
        sMem1[r.z] = key.z;
        sMem1[r.w] = key.w;
        __syncthreads();

        // This access has 4-way bank conflicts
        key = sMem[threadIdx.x];
#endif

        __syncthreads();
    }
}

//----------------------------------------------------------------------------
//
// radixSortBlocks sorts all blocks of data independently in shared
// memory.  Each thread block (CTA) sorts one block of 4*CTA_SIZE elements
//
// The radix sort is done in two stages.  This stage calls radixSortBlock on each 
// block independently, sorting on the basis of bits (startbit) -> (startbit + nbits)
// 
// Template parameters are used to generate efficient code for various special cases
// For example, we have to handle arrays that are a multiple of the block size (fullBlocks)
// differently than arrays that are not.  "flip" is used to only compile in the
// float flip code when float keys are used.  "loop" is used when persistent CTAs
// are used. 
//
// By persistent CTAs we mean that we launch only as many thread blocks as can 
// be resident in the GPU and no more, rather than launching as many threads as
// we have elements. Persistent CTAs loop over blocks of elements until all work
// is complete.  This can be faster in some cases.  In our tests it is faster
// for large sorts (and the threshold is higher on compute version 1.1 and earlier
// GPUs than it is on compute version 1.2 GPUs.
//----------------------------------------------------------------------------
    __extern__shared__ uint4 sMem[];
template<uint nbits, uint startbit, bool fullBlocks, bool flip, bool loop>
__global__ void radixSortBlocksKeysOnly(uint4* keysOut, uint4* keysIn, uint numElements, uint totalBlocks)
{

    uint4 key;

    uint blockId = blockIdx.x;
    
    while (!loop || blockId < totalBlocks)
    {
        uint i = blockId * blockDim.x + threadIdx.x;
        uint idx = i << 2;

        // handle non-full last block if array is not multiple of 1024 numElements
        if (!fullBlocks && idx+3 >= numElements)
        {
            if (idx >= numElements)
            {
                key   = make_uint4(UINT_MAX, UINT_MAX, UINT_MAX, UINT_MAX);
            }
            else
            {
                // for non-full block, we handle uint1 values instead of uint4
                uint *keys1    = (uint*)keysIn;

                key.x = (idx   < numElements) ? floatFlip<flip>(keys1[idx])   : UINT_MAX;
                key.y = (idx+1 < numElements) ? floatFlip<flip>(keys1[idx+1]) : UINT_MAX;
                key.z = (idx+2 < numElements) ? floatFlip<flip>(keys1[idx+2]) : UINT_MAX;
                key.w = UINT_MAX;
            }
        }
        else
        {
            key = keysIn[i];
            if (flip)
            {
                key.x = floatFlip<flip>(key.x);
                key.y = floatFlip<flip>(key.y);
                key.z = floatFlip<flip>(key.z);
                key.w = floatFlip<flip>(key.w);
            }            
        }
        __syncthreads();
        radixSortBlockKeysOnly<nbits, startbit>(key);

        // handle non-full last block if array is not multiple of 1024 numElements
        if(!fullBlocks && idx+3 >= numElements)
        {
            if (idx < numElements) 
            {
                // for non-full block, we handle uint1 values instead of uint4
                uint *keys1   = (uint*)keysOut;

                keys1[idx]   = key.x;

                if (idx + 1 < numElements)
                {
                    keys1[idx + 1]   = key.y;

                    if (idx + 2 < numElements)
                    {
                        keys1[idx + 2]   = key.z;
                    }
                }
            }
        }
        else
        {
            keysOut[i]   = key;
        }

        if (loop)
            blockId += gridDim.x;
        else
            break;
    }
}

//----------------------------------------------------------------------------
// reorderData shuffles data in the array globally after the radix offsets 
// have been found. On compute version 1.1 and earlier GPUs, this code depends 
// on RadixSort::CTA_SIZE being 16 * number of radices (i.e. 16 * 2^nbits).
// 
// On compute version 1.1 GPUs ("manualCoalesce=true") this function ensures
// that all writes are coalesced using extra work in the kernel.  On later
// GPUs coalescing rules have been relaxed, so this extra overhead hurts 
// performance.  On these GPUs we set manualCoalesce=false and directly store
// the results.
//
// Template parameters are used to generate efficient code for various special cases
// For example, we have to handle arrays that are a multiple of the block size 
// (fullBlocks) differently than arrays that are not.  "loop" is used when persistent 
// CTAs are used. 
//
// By persistent CTAs we mean that we launch only as many thread blocks as can 
// be resident in the GPU and no more, rather than launching as many threads as
// we have elements. Persistent CTAs loop over blocks of elements until all work
// is complete.  This can be faster in some cases.  In our tests it is faster
// for large sorts (and the threshold is higher on compute version 1.1 and earlier
// GPUs than it is on compute version 1.2 GPUs.
//----------------------------------------------------------------------------
template<uint startbit, bool fullBlocks, bool manualCoalesce, bool unflip, bool loop>
__global__ void reorderDataKeysOnly(uint  *outKeys, 
                                    uint2 *keys, 
                                    uint  *blockOffsets, 
                                    uint  *offsets, 
                                    uint  *sizes, 
                                    uint   numElements,
                                    uint   totalBlocks)
{
    __shared__ uint2 sKeys2[RadixSort::CTA_SIZE];
    __shared__ uint sOffsets[16];
    __shared__ uint sBlockOffsets[16];

    uint *sKeys1   = (uint*)sKeys2; 

    uint blockId = blockIdx.x;

    while (!loop || blockId < totalBlocks)
    {
        uint i = blockId * blockDim.x + threadIdx.x;

        // handle non-full last block if array is not multiple of 1024 numElements
        if(!fullBlocks && (((i + 1) << 1) > numElements))
        {
            uint *keys1   = (uint*)keys;
            uint j = i << 1; 

            sKeys1[threadIdx.x << 1]   = (j < numElements) ? keys1[j]   : UINT_MAX; 
            j++; 
            sKeys1[(threadIdx.x << 1) + 1]   = (j < numElements) ? keys1[j]   : UINT_MAX; 
        }
        else
        {
            sKeys2[threadIdx.x]   = keys[i];
        }

        if (!manualCoalesce)
        {
            if(threadIdx.x < 16)  
            {
                sOffsets[threadIdx.x]      = offsets[threadIdx.x * totalBlocks + blockId];
                sBlockOffsets[threadIdx.x] = blockOffsets[blockId * 16 + threadIdx.x];
            }
            __syncthreads();

            uint radix = (sKeys1[threadIdx.x] >> startbit) & 0xF;
            uint globalOffset = sOffsets[radix] + threadIdx.x - sBlockOffsets[radix];

            if (fullBlocks || globalOffset < numElements)
            {
                outKeys[globalOffset]   = floatUnflip<unflip>(sKeys1[threadIdx.x]);
            }

            radix = (sKeys1[threadIdx.x + RadixSort::CTA_SIZE] >> startbit) & 0xF;
            globalOffset = sOffsets[radix] + threadIdx.x + RadixSort::CTA_SIZE - sBlockOffsets[radix];

            if (fullBlocks || globalOffset < numElements)
            {
                outKeys[globalOffset]   = floatUnflip<unflip>(sKeys1[threadIdx.x + RadixSort::CTA_SIZE]);
            }
        }
        else
        {
            __shared__ uint sSizes[16];

            if(threadIdx.x < 16)  
            {
                sOffsets[threadIdx.x]      = offsets[threadIdx.x * totalBlocks + blockId];
                sBlockOffsets[threadIdx.x] = blockOffsets[blockId * 16 + threadIdx.x];
                sSizes[threadIdx.x]        = sizes[threadIdx.x * totalBlocks + blockId];
            }
            __syncthreads();

            // 1 half-warp is responsible for writing out all values for 1 radix. 
            // Loops if there are more than 16 values to be written out. 
            // All start indices are rounded down to the nearest multiple of 16, and
            // all end indices are rounded up to the nearest multiple of 16.
            // Thus it can do extra work if the start and end indices are not multiples of 16
            // This is bounded by a factor of 2 (it can do 2X more work at most).

            const uint halfWarpID     = threadIdx.x >> 4;

            const uint halfWarpOffset = threadIdx.x & 0xF;
            const uint leadingInvalid = sOffsets[halfWarpID] & 0xF;

            uint startPos = sOffsets[halfWarpID] & 0xFFFFFFF0;
            uint endPos   = (sOffsets[halfWarpID] + sSizes[halfWarpID]) + 15 - 
                ((sOffsets[halfWarpID] + sSizes[halfWarpID] - 1) & 0xF);
            uint numIterations = endPos - startPos;

            uint outOffset = startPos + halfWarpOffset;
            uint inOffset  = sBlockOffsets[halfWarpID] - leadingInvalid + halfWarpOffset;

            for(uint j = 0; j < numIterations; j += 16, outOffset += 16, inOffset += 16)
            {       
                if( (outOffset >= sOffsets[halfWarpID]) && 
                    (inOffset - sBlockOffsets[halfWarpID] < sSizes[halfWarpID])) 
                {
                    if(blockId < totalBlocks - 1 || outOffset < numElements) 
                    {
                        outKeys[outOffset] = floatUnflip<unflip>(sKeys1[inOffset]);
                    }
                }       
            }
        }

        if (loop)
        {
            blockId += gridDim.x;
            __syncthreads();
        }
        else
            break;
    }
}

//----------------------------------------------------------------------------
// Perform one step of the radix sort.  Sorts by nbits key bits per step, 
// starting at startbit.
//----------------------------------------------------------------------------
template<uint nbits, uint startbit, bool flip, bool unflip>
void radixSortStepKeysOnly(uint *keys, 
                   uint *tempKeys, 
                   uint *counters, 
                   uint *countersSum, 
                   uint *blockOffsets, 
                   CUDPPHandle scanPlan,
                   uint numElements)
{
    const uint eltsPerBlock = RadixSort::CTA_SIZE * 4;
    const uint eltsPerBlock2 = RadixSort::CTA_SIZE * 2;

    bool fullBlocks = ((numElements % eltsPerBlock) == 0);
    uint numBlocks = (fullBlocks) ? 
        (numElements / eltsPerBlock) : 
        (numElements / eltsPerBlock + 1);
    uint numBlocks2 = ((numElements % eltsPerBlock2) == 0) ?
        (numElements / eltsPerBlock2) : 
        (numElements / eltsPerBlock2 + 1);

    bool loop = numBlocks > 65535;
    //bool loop2 = numBlocks2 > 65535;
    uint blocks = loop ? 65535 : numBlocks;
    uint blocksFind = loop ? 65535 : numBlocks2;
    uint blocksReorder = loop ? 65535 : numBlocks2;

    uint threshold = fullBlocks ? persistentCTAThresholdFullBlocks[1] : persistentCTAThreshold[1];

    if (numElements >= threshold)
    {
        loop = (numElements > 262144) || (numElements >= 32768 && numElements < 65536);
        //loop2 = (numElements > 262144) || (numElements >= 32768 && numElements < 65536);
        blocks = numBlocks;//loop ? numCTAs[SORT_KERNEL_RADIX_SORT_BLOCKS] : numBlocks;
        blocksFind = numBlocks2;//loop ? numCTAs[SORT_KERNEL_FIND_RADIX_OFFSETS] : numBlocks2;
        blocksReorder = numBlocks2;//loop ? numCTAs[SORT_KERNEL_REORDER_DATA] : numBlocks2;
    }
  
    if (fullBlocks)
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocks = flip ? numCTAs(radixSortBlocksKeysOnly<4, 0, true, true, true>) : 
		       	        numCTAs(radixSortBlocksKeysOnly<4, 0, true, false, true>);
            }
        
            radixSortBlocksKeysOnly<nbits, startbit, true, flip, true>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)keys, numElements, numBlocks);
        }
        else
            radixSortBlocksKeysOnly<nbits, startbit, true, flip, false>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)keys, numElements, numBlocks);
    }
    else
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocks = flip ? numCTAs(radixSortBlocksKeysOnly<4, 0, false, true, true>) : 
		       	        numCTAs(radixSortBlocksKeysOnly<4, 0, false, false, true>);
            }

            radixSortBlocksKeysOnly<nbits, startbit, false, flip, true>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)keys, numElements, numBlocks);
        }
        else
            radixSortBlocksKeysOnly<nbits, startbit, false, flip, false>
                <<<blocks, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)tempKeys, (uint4*)keys, numElements, numBlocks);

    }

    if (fullBlocks)
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocksFind = numCTAs(findRadixOffsets<0, true, true>);
            }
            findRadixOffsets<startbit, true, true>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
        }
        else
            findRadixOffsets<startbit, true, false>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
    }
    else
    {
        if (loop)
        {
            if (numElements >= threshold) 
            {
                blocksFind = numCTAs(findRadixOffsets<0, false, true>);
            }
            findRadixOffsets<startbit, false, true>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);
        }
        else
            findRadixOffsets<startbit, false, false>
                <<<blocksFind, RadixSort::CTA_SIZE, 3 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint2*)tempKeys, counters, blockOffsets, numElements, numBlocks2);

    }

    //cudppScan(scanPlan, countersSum, counters, 16*numBlocks2);

    if (fullBlocks)
    {
        if (bManualCoalesce)
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ? 
		        numCTAs(reorderDataKeysOnly<0, true, true, true, true>) : 
                        numCTAs(reorderDataKeysOnly<0, true, true, false, true>);
                }
                reorderDataKeysOnly<startbit, true, true, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                     numElements, numBlocks2);
            }
            else
                reorderDataKeysOnly<startbit, true, true, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                     numElements, numBlocks2);
        }
        else
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ?
                        numCTAs(reorderDataKeysOnly<0, true, false, true, true>) :
                        numCTAs(reorderDataKeysOnly<0, true, false, false, true>);
                }
                reorderDataKeysOnly<startbit, true, false, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                    numElements, numBlocks2);
            }
            else
                reorderDataKeysOnly<startbit, true, false, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                    numElements, numBlocks2);
        }
    }
    else
    {
        if (bManualCoalesce)
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ? 
                        numCTAs(reorderDataKeysOnly<0, false, true, true, true>) :
                        numCTAs(reorderDataKeysOnly<0, false, true, false, true>);
                }
                reorderDataKeysOnly<startbit, false, true, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                     numElements, numBlocks2);
            }
            else
                reorderDataKeysOnly<startbit, false, true, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                     numElements, numBlocks2);
        }
        else
        {
            if (loop)
            {
                if (numElements >= threshold) 
                {
                    blocksReorder = unflip ?
                        numCTAs(reorderDataKeysOnly<0, false, false, true, true>) :
                        numCTAs(reorderDataKeysOnly<0, false, false, false, true>);
                }
                reorderDataKeysOnly<startbit, false, false, unflip, true>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                     numElements, numBlocks2);
            }
            else
                reorderDataKeysOnly<startbit, false, false, unflip, false>
                    <<<blocksReorder, RadixSort::CTA_SIZE>>>
                    (keys, (uint2*)tempKeys, blockOffsets, countersSum, counters, 
                    numElements, numBlocks2);
        }
    }

    checkCudaError("radixSortStepKeysOnly");
}

//----------------------------------------------------------------------------
// Optimization for sorts of fewer than 4 * CTA_SIZE elements
//----------------------------------------------------------------------------
template <bool flip>
void radixSortSingleBlockKeysOnly(uint *keys, 
                                  uint numElements)
{
    bool fullBlocks = (numElements % (RadixSort::CTA_SIZE * 4) == 0);
    if (fullBlocks)
    {
        radixSortBlocksKeysOnly<32, 0, true, flip, false>
            <<<1, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)keys, (uint4*)keys, numElements, 1 );
    }
    else
    {
        radixSortBlocksKeysOnly<32, 0, false, flip, false>
            <<<1, RadixSort::CTA_SIZE, 4 * RadixSort::CTA_SIZE * sizeof(uint)>>>
                ((uint4*)keys, (uint4*)keys, numElements, 1 );
    }

    if (flip)
        unflipFloats<<<1, RadixSort::CTA_SIZE>>>(keys, numElements);


    checkCudaError("radixSortSingleBlock");
}

//----------------------------------------------------------------------------
// Optimization for sorts of WARP_SIZE or fewer elements
//----------------------------------------------------------------------------
template <bool flip>
__global__ 
void radixSortSingleWarpKeysOnly(uint *keys, 
                                 uint numElements)
{
    volatile __shared__ uint sKeys[RadixSort::WARP_SIZE];
    volatile __shared__ uint sFlags[RadixSort::WARP_SIZE];

    sKeys[threadIdx.x]   = floatFlip<flip>(keys[threadIdx.x]);
    
    __SYNC // emulation only

    for(uint i = 1; i < numElements; i++)
    {
        uint key_i = sKeys[i];
        
        sFlags[threadIdx.x] = 0;
        
        if( (threadIdx.x < i) && (sKeys[threadIdx.x] > key_i) ) 
        {
            uint temp = sKeys[threadIdx.x];
            sFlags[threadIdx.x] = 1;
            sKeys[threadIdx.x + 1] = temp;
            sFlags[threadIdx.x + 1] = 0;
        }
        if(sFlags[threadIdx.x] == 1 )
        {
            sKeys[threadIdx.x] = key_i;
        }

        __SYNC // emulation only

    }
    keys[threadIdx.x]   = floatUnflip<flip>(sKeys[threadIdx.x]);
}

//----------------------------------------------------------------------------
// Main key-only radix sort function.  Sorts in place in the keys and values 
// arrays, but uses the other device arrays as temporary storage.  All pointer 
// parameters are device pointers.  Uses cudppScan() for the prefix sum of
// radix counters.
//----------------------------------------------------------------------------
extern "C"
void radixSortKeysOnly(uint *keys, 
                       uint *tempKeys, 
                       uint *counters,
                       uint *countersSum,
                       uint *blockOffsets,
                       CUDPPHandle scanPlan,
                       uint numElements, 
                       uint keyBits,
                       bool flipBits = false)
{
    if(numElements <= RadixSort::WARP_SIZE)
    {
        printf("numElements <= RadixSort::WARP_SIZE \n");
        if (flipBits)
            radixSortSingleWarpKeysOnly<true><<<1, numElements>>>(keys, numElements);
        else
            radixSortSingleWarpKeysOnly<false><<<1, numElements>>>(keys, numElements);
        checkCudaError("radixSortSingleWarp");
        return;
    }
    if(numElements <= RadixSort::CTA_SIZE * 4)
    {
        printf("numElements <= RadixSort::CTA_SIZE * 4 \n");
        if (flipBits)
            radixSortSingleBlockKeysOnly<true>(keys, numElements);
        else
            radixSortSingleBlockKeysOnly<false>(keys, numElements);
        return;
    }

    // flip float bits on the first pass, unflip on the last pass
    if (flipBits) 
    {
            printf("flipBits true \n");
            radixSortStepKeysOnly<4,  0, true, false>(keys, tempKeys, 
                                                      counters, countersSum, blockOffsets, 
                                                      scanPlan, numElements);
    }
    else
    {
            printf("flipBits false \n");
            radixSortStepKeysOnly<4,  0, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }

    if (keyBits > 4)
    {
            printf("keyBits > 4 \n");
            radixSortStepKeysOnly<4,  4, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }
    if (keyBits > 8)
    {
            printf("keyBits > 8 \n");
            radixSortStepKeysOnly<4,  8, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }
    if (keyBits > 12)
    {
            printf("keyBits > 12 \n");
            radixSortStepKeysOnly<4, 12, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }
    if (keyBits > 16)
    {
            printf("keyBits > 16 \n");
            radixSortStepKeysOnly<4, 16, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }
    if (keyBits > 20)
    {
            printf("keyBits > 20 \n");
            radixSortStepKeysOnly<4, 20, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }
    if (keyBits > 24)
    {
            printf("keyBits > 24 \n");
            radixSortStepKeysOnly<4, 24, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
    }
    if (keyBits > 28)
    {
        if (flipBits) // last pass
        {
            radixSortStepKeysOnly<4, 28, false, true>(keys, tempKeys, 
                                                      counters, countersSum, blockOffsets, 
                                                      scanPlan, numElements);
        }
        else
        {
            radixSortStepKeysOnly<4, 28, false, false>(keys, tempKeys, 
                                                       counters, countersSum, blockOffsets, 
                                                       scanPlan, numElements);
        }
    }

    checkCudaError("radixSortKeysOnly");
}

//----------------------------------------------------------------------------
// Main float key-only radix sort function.  Sorts in place in the keys and values 
// arrays, but uses the other device arrays as temporary storage.  All pointer 
// parameters are device pointers.  Uses cudppScan() for the prefix sum of
// radix counters.
//----------------------------------------------------------------------------
extern "C"
void radixSortFloatKeysOnly(float *keys, 
                            float *tempKeys, 
                            uint  *counters,
                            uint  *countersSum,
                            uint  *blockOffsets,
                            CUDPPHandle scanPlan,
                            uint  numElements, 
                            uint  keyBits,
                            bool  negativeKeys)
{
    radixSortKeysOnly((uint*)keys, (uint*)tempKeys, counters, countersSum, blockOffsets, 
                       scanPlan, numElements, keyBits, negativeKeys);
    checkCudaError("radixSortFloatKeys");
}

void makeRandomUintVector(unsigned int *a, unsigned int numElements, unsigned int keybits)
{
    // Fill up with some random data
    int keyshiftmask = 0;
    if (keybits > 16) keyshiftmask = (1 << (keybits - 16)) - 1;
    int keymask = 0xffff;
    if (keybits < 16) keymask = (1 << keybits) - 1;

    srand(95123);
    for(unsigned int i=0; i < numElements; ++i)
    {
        a[i] = ((rand() & keyshiftmask)<<16) | (rand() & keymask);
    }
}

//***********************************************************************
//! The Driver
//***********************************************************************

int main() {
  int keybits = 32;

  // Set numElements as your wish 
  unsigned int numElements = 32;
  
  //bool keysOnly = false;
  bool keysOnly = true;

  unsigned int *h_keys       = (unsigned int*)malloc(numElements*sizeof(unsigned int));
  unsigned int *h_keysSorted = (unsigned int *)malloc(numElements*sizeof(unsigned int));
  unsigned int *h_values     = 0;
  if (!keysOnly)
      h_values = (unsigned int *)malloc(numElements*sizeof(unsigned int));

  // Fill up with some random data
  makeRandomUintVector((unsigned int*)h_keys, numElements, keybits);

  if (!keysOnly)
  {
      for(unsigned int i=0; i < numElements; ++i)
      {
          h_values[i] = i;
      }
  }

  // Copy data onto the GPU
  unsigned int *d_keys;
  unsigned int *d_values;
  hipMalloc((void **)&d_keys, numElements*sizeof(unsigned int));
  if (!keysOnly)
      hipMalloc((void **)&d_values, numElements*sizeof(unsigned int));
  else
      d_values = 0;

  // Creat the RadixSort object
  RadixSort radixsort(numElements, keysOnly);

  hipMemcpy(d_keys, h_keys, numElements * sizeof(unsigned int), hipMemcpyHostToDevice);
  if (!keysOnly)
     hipMemcpy(d_values, h_values, numElements * sizeof(unsigned int), hipMemcpyHostToDevice);

  // reset data before sort
  hipMemcpy(d_keys, h_keys, numElements * sizeof(unsigned int), hipMemcpyHostToDevice);
  if (!keysOnly)
     hipMemcpy(d_values, h_values, numElements * sizeof(unsigned int), hipMemcpyHostToDevice);

  // unsigned int sort 
  radixsort.sort((unsigned int*)d_keys, d_values, numElements, keybits);

  // Get results back to host for correctness checking
  hipMemcpy(h_keysSorted, d_keys, numElements * sizeof(unsigned int), hipMemcpyDeviceToHost);

  if (!keysOnly)
      hipMemcpy(h_values, d_values, numElements * sizeof(unsigned int), hipMemcpyDeviceToHost);
  else
      h_values = 0;

#ifndef _SYM
  // post conditions
  for (int i = 1; i < numElements; i++) {
    if (h_keysSorted[i] < h_keysSorted[i-1]) {
      printf("The sorting algorithm is incorrect since keys[%d] < keys[%d]!", i, i-1);
    }
  }
#endif

  hipFree(d_keys);
  hipFree(d_values);
  free(h_keys);
  free(h_values);
}
