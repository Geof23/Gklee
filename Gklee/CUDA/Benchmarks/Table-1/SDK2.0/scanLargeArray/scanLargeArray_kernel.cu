#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifndef _SCAN_BEST_KERNEL_CU_
#define _SCAN_BEST_KERNEL_CU_

#include <stdio.h>

// Define this to more rigorously avoid bank conflicts, 
// even at the lower (root) levels of the tree
// Note that due to the higher addressing overhead, performance 
// is lower with ZERO_BANK_CONFLICTS enabled.  It is provided
// as an example.
//#define ZERO_BANK_CONFLICTS 

// 16 banks on G80
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// https://www.cs.unc.edu/~prins/Classes/633/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// excellent paper "Prefix sums and their applications".
// http://www.cs.cmu.edu/~blelloch/papers/Ble93.pdf
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//

template <bool isNP2>
__device__ void loadSharedChunkFromMem(float *s_data,
                                       const float *g_idata, 
                                       int n, int baseIndex,
                                       int& ai, int& bi, 
                                       int& mem_ai, int& mem_bi, 
                                       int& bankOffsetA, int& bankOffsetB)
{
    int thid = threadIdx.x;
    mem_ai = baseIndex + threadIdx.x;
    mem_bi = mem_ai + blockDim.x;

    ai = thid;
    bi = thid + blockDim.x;

    // compute spacing to avoid bank conflicts
    bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    // pad values beyond n with zeros
    printf("ai + bankOffsetA: %d, mem_ai: %d \n", ai + bankOffsetA, mem_ai);
    s_data[ai + bankOffsetA] = g_idata[mem_ai]; 
    
    if (isNP2) // compile-time decision
    {
        s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0; 
    }
    else
    {
        s_data[bi + bankOffsetB] = g_idata[mem_bi]; 
    }
}

template <bool isNP2>
__device__ void storeSharedChunkToMem(float* g_odata, 
                                      const float* s_data,
                                      int n, 
                                      int ai, int bi, 
                                      int mem_ai, int mem_bi,
                                      int bankOffsetA, int bankOffsetB)
{
    __syncthreads();

    // write results to global memory
    g_odata[mem_ai] = s_data[ai + bankOffsetA]; 
    if (isNP2) // compile-time decision
    {
        if (bi < n)
            g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
    else
    {
        g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
}

template <bool storeSum>
__device__ void clearLastElement(float* s_data, 
                                 float *g_blockSums, 
                                 int blockIndex)
{
    if (threadIdx.x == 0)
    {
        int index = (blockDim.x << 1) - 1;
        index += CONFLICT_FREE_OFFSET(index);
        
        if (storeSum) // compile-time decision
        {
            // write this block's total sum to the corresponding index in the blockSums array
            g_blockSums[blockIndex] = s_data[index];
        }

        // zero the last element in the scan so it will propagate back to the front
        s_data[index] = 0;
    }
}



__device__ unsigned int buildSum(float *s_data)
{
    unsigned int thid = threadIdx.x;
    unsigned int stride = 1;
    
    // build the sum in place up the tree
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    return stride;
}

__device__ void scanRootToLeaves(float *s_data, unsigned int stride)
{
     unsigned int thid = threadIdx.x;

    // traverse down the tree building the scan in place
    for (int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
}

template <bool storeSum>
__device__ void prescanBlock(float *data, int blockIndex, float *blockSums)
{
    int stride = buildSum(data);               // build the sum in place up the tree
    clearLastElement<storeSum>(data, blockSums, 
                               (blockIndex == 0) ? blockIdx.x : blockIndex);
    scanRootToLeaves(data, stride);            // traverse down tree to build the scan 
}

__extern__shared__ float s_data[];

template <bool storeSum, bool isNP2>
__global__ void prescan(float *g_odata, 
                        const float *g_idata, 
                        float *g_blockSums, 
                        int n, 
                        int blockIndex, 
                        int baseIndex)
{
    int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;

    // load data into shared memory
    loadSharedChunkFromMem<isNP2>(s_data, g_idata, n, 
                                  (baseIndex == 0) ? 
                                  __mul24(blockIdx.x, (blockDim.x << 1)):baseIndex,
                                  ai, bi, mem_ai, mem_bi, 
                                  bankOffsetA, bankOffsetB); 
    // scan the data in each block
    prescanBlock<storeSum>(s_data, blockIndex, g_blockSums); 
    // write results to device memory
    storeSharedChunkToMem<isNP2>(g_odata, s_data, n, 
                                 ai, bi, mem_ai, mem_bi, 
                                 bankOffsetA, bankOffsetB);  
}


__global__ void uniformAdd(float *g_data, 
                           float *uniforms, 
                           int n, 
                           int blockOffset, 
                           int baseIndex)
{
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    printf("baseIndex: %d \n", baseIndex);
    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x; 

    __syncthreads();
    
    // note two adds per thread
    g_data[address]              += uni;
    g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}


#endif // #ifndef _SCAN_BEST_KERNEL_CU_

