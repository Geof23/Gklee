#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifndef _SCAN_NAIVE_KERNEL_H_
#define _SCAN_NAIVE_KERNEL_H_

///////////////////////////////////////////////////////////////////////////////
//! Naive compute implementation of scan, one thread per element
//! Not work efficient: log(n) steps, but n * (log(n) - 1) adds.
//! Not shared storage efficient either -- this requires ping-ponging
//! arrays in shared memory due to hazards so 2 * n storage space.
//!
//! Pro: Simple
//! Con: Not work efficient
//!
//! @param g_odata  output data in global memory
//! @param g_idata  input data in global memory
//! @param n        input number of elements to scan from input data
///////////////////////////////////////////////////////////////////////////////
   __extern__shared__  float temp[];
__global__ void scan_naive(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels

    int thid = threadIdx.x;

    int pout = 0;
    int pin = 1;

    // Cache the computational window in shared memory
    temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;

    for (int offset = 1; offset < n; offset *= 2)
    {
        pout = 1 - pout;
        pin  = 1 - pout;
        __syncthreads();

        temp[pout*n+thid] = temp[pin*n+thid];

        if (thid >= offset)
            temp[pout*n+thid] += temp[pin*n+thid - offset];
    }

    __syncthreads();

    g_odata[thid] = temp[pout*n+thid];
}

#endif // #ifndef _SCAN_NAIVE_KERNEL_H_
