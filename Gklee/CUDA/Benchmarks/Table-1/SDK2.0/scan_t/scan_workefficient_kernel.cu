#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifndef _SCAN_WORKEFFICIENT_KERNEL_H_
#define _SCAN_WORKEFFICIENT_KERNEL_H_

///////////////////////////////////////////////////////////////////////////////
//! Work-efficient compute implementation of scan, one thread per 2 elements
//! Work-efficient: O(log(n)) steps, and O(n) adds.
//! Also shared storage efficient: Uses n elements in shared mem -- no ping-ponging
//! Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
//! and Their Applications", or Prins and Chatterjee PRAM course notes:
//! https://www.cs.unc.edu/~prins/Classes/633/Handouts/pram.pdf
//!
//! Pro: Work Efficient
//! Con: Shared memory bank conflicts due to the addressing used.
//
//! @param g_odata  output data in global memory
//! @param g_idata  input data in global memory
//! @param n        input number of elements to scan from input data
///////////////////////////////////////////////////////////////////////////////
    __extern__shared__  float temp[];
__global__ void scan_workefficient(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels

    int thid = threadIdx.x;

    int offset = 1;

    // Cache the computational window in shared memory
    temp[2*thid]   = g_idata[2*thid];
    temp[2*thid+1] = g_idata[2*thid+1];

    // build the sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi] += temp[ai];
        }

        offset *= 2;
    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        temp[n - 1] = 0;
    }   

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            float t   = temp[ai];
            temp[ai]  = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    // write results to global memory
    g_odata[2*thid]   = temp[2*thid];
    g_odata[2*thid+1] = temp[2*thid+1];
}

#endif // #ifndef _SCAN_WORKEFFICIENT_KERNEL_H_
