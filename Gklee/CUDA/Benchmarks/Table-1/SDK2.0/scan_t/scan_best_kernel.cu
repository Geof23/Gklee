#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifndef _SCAN_BEST_KERNEL_H_
#define _SCAN_BEST_KERNEL_H_

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

// Define this to more rigorously avoid bank conflicts, even at the lower (root) levels of the tree
//#define ZERO_BANK_CONFLICTS 

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

#ifdef CHECK_BANK_CONFLICTS
#define TEMP(index)   cutilBankChecker(temp, index)
#else
#define TEMP(index)   temp[index]
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// https://www.cs.unc.edu/~prins/Classes/633/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// Excellent paper "Prefix sums and their applications".
// http://www.cs.cmu.edu/~blelloch/papers/Ble93.pdf
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//
// @param g_odata  output data in global memory
// @param g_idata  input data in global memory
// @param n        input number of elements to scan from input data
__extern__shared__  float temp[];

__global__ void scan_best(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels

    int thid = threadIdx.x;

    int ai = thid;
    int bi = thid + (n/2);

    // compute spacing to avoid bank conflicts
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    TEMP(ai + bankOffsetA) = g_idata[ai]; 
    TEMP(bi + bankOffsetB) = g_idata[bi]; 

    int offset = 1;

    printf("The n in scan_best: %d \n", n);
    // build the sum in place up the tree
    for (int d = n/2; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            TEMP(bi) += TEMP(ai);
        }

        offset *= 2;
    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        int index = n - 1;
        index += CONFLICT_FREE_OFFSET(index);
        TEMP(index) = 0;
    }   

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset /= 2;

        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = TEMP(ai);
            TEMP(ai) = TEMP(bi);
            TEMP(bi) += t;
        }
    }

    __syncthreads();

    // write results to global memory
    g_odata[ai] = TEMP(ai + bankOffsetA); 
    g_odata[bi] = TEMP(bi + bankOffsetB); 
}
#endif // #ifndef _SCAN_BEST_KERNEL_H_

