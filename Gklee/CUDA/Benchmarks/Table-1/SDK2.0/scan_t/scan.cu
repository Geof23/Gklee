#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
//#include <cutil_inline.h>

// includes, kernels
//#include <scan_naive_kernel.cu>
//#include <scan_workefficient_kernel.cu>
//#include <scan_best_kernel.cu>
#include "scan_naive_kernel.cu"
#include "scan_workefficient_kernel.cu"
#include "scan_best_kernel.cu"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

// regression test functionality
extern "C" 
unsigned int compare( const float* reference, const float* data, 
                      const unsigned int len);
extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len)
{
  reference[0] = 0;
  double total_sum = 0;
  unsigned int i;
  for( i = 1; i < len; ++i) 
  {
      total_sum += idata[i-1];
      reference[i] = idata[i-1] + reference[i-1];
  }
  // Here it should be okay to use != because we have integer values
  // in a range where float can be exactly represented
  if (total_sum != reference[i-1])
      printf("Warning: exceeding single-precision accuracy.  Scan will be inaccurate.\n");
  
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    //cutilExit(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
    //    cutilDeviceInit(argc, argv);
    //else
    //    hipSetDevice( cutGetMaxGflopsDeviceId() );

    unsigned int num_elements = 512;
    //cutGetCmdLineArgumenti( argc, (const char**) argv, "n", (int*)&num_elements);

    //unsigned int timer;
    //cutilCheckError( cutCreateTimer(&timer));
    
    const unsigned int num_threads = num_elements / 2;
    const unsigned int mem_size = sizeof( float) * num_elements;

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extra_space = num_elements / NUM_BANKS;

#ifdef ZERO_BANK_CONFLICTS
    extra_space += extra_space / NUM_BANKS;
#endif

    const unsigned int shared_mem_size = sizeof(float) * 
        (num_elements + extra_space);

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( mem_size);
      
    // initialize the input data on the host to be integer values
    // between 0 and 1000

#ifndef _SYM
    for( unsigned int i = 0; i < num_elements; ++i) 
    {
        h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
    }
    // compute reference solution
    float* reference = (float*) malloc( mem_size);  
#else
    klee_make_symbolic(h_data, mem_size, "h_data_input");
#endif

    // allocate device memory input and output arrays
    float* d_idata;
    float* d_odata[3];
    hipMalloc( (void**) &d_idata, mem_size);
    hipMalloc( (void**) &(d_odata[0]), mem_size);
    hipMalloc( (void**) &(d_odata[1]), mem_size);
    hipMalloc( (void**) &(d_odata[2]), mem_size);

    // copy host memory to device input array
    //cutilSafeCall( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );
    hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice);

    // setup execution parameters
    // Note that these scans only support a single thread-block worth of data,
    // but we invoke them here on many blocks so that we can accurately compare
    // performance
#ifndef __DEVICE_EMULATION__
    dim3  grid(256, 1, 1);  
#else
    dim3  grid(1, 1, 1); // only one run block in device emu mode or it will be too slow
#endif
    dim3  threads(num_threads*2, 1, 1);

    // make sure there are no CUDA errors before we start
    //cutilCheckMsg("Kernel execution failed");
    printf("Kernel execution failed \n");

    printf("Running parallel prefix sum (scan) of %d elements\n", num_elements);
    printf("Comparing 3 versions:\n\n");
  
    // execute the kernels
    //unsigned int numIterations = 100;
    unsigned int numIterations = 1;
    printf("1. scan_naive -- not work efficient (O(n log n) adds).\n");
    //cutStartTimer(timer);
    for (unsigned int i = 0; i < numIterations; ++i)
    {
        scan_naive<<< grid, threads, 2 * shared_mem_size >>>
            (d_odata[0], d_idata, num_elements);
    }
    //hipDeviceSynchronize();
    //cutStopTimer(timer);
    //printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    //cutResetTimer(timer);

    threads.x /= 2;
    
    printf("2. scan_workefficient -- Work efficient (O(n) adds), but many bank conflicts.\n");
    //cutStartTimer(timer);
    for (unsigned int i = 0; i < numIterations; ++i)
    {
        scan_workefficient<<< grid, threads, shared_mem_size >>>
            (d_odata[1], d_idata, num_elements);
    }
    //hipDeviceSynchronize();
    //cutStopTimer(timer);
    //printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    //cutResetTimer(timer);
    printf("3. scan_best -- work efficient with very few bank conflicts.\n");
    //cutStartTimer(timer);
    for (unsigned int i = 0; i < numIterations; ++i)
    {
        scan_best<<< grid, threads, shared_mem_size >>>
            (d_odata[2], d_idata, num_elements);
    }
    //hipDeviceSynchronize();
    //cutStopTimer(timer);
    //printf("Average time: %f ms\n\n", cutGetTimerValue(timer) / numIterations);
    //cutResetTimer(timer);

    // check for any errors
    //cutilCheckMsg("Kernel execution failed");
    printf("Kernel execution failed \n");

    for (int i = 0; i < 3; ++i) // check all 3 results
    {
        // copy result from device to host
        //cutilSafeCall(hipMemcpy( h_data, d_odata[i], sizeof(float) * num_elements, 
        //                           hipMemcpyDeviceToHost));
        hipMemcpy( h_data, d_odata[i], sizeof(float) * num_elements, 
                    hipMemcpyDeviceToHost);

        // If this is a regression test write the results to a file
        //if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
        //{
            // write file for regression test 
            //cutWriteFilef( "./data/result.dat", h_data, num_elements, 0.0);
        //}
        //else 
        //{
            // custom output handling when no regression test running
            // in this case check if the result is equivalent to the expected soluion
            
            // We can use an epsilon of 0 since values are integral and in a range 
            // that can be exactly represented
            //float epsilon = 0.0f;
            //unsigned int result_regtest = cutComparefe( reference, h_data, num_elements, epsilon);
            //char* names[] = {"scan_naive", "scan_workefficient", "scan_best"};
            //printf( "%s: Test %s\n", names[i], (1 == result_regtest) ? "PASSED" : "FAILED");
        //}
    }

    printf("\nCheck out the CUDA Data Parallel Primitives Library for more on scan.\n");
    printf("http://www.gpgpu.org/developer/cudpp\n");

    // cleanup memory
    free( h_data);
#ifndef _SYM
    free( reference);
#endif
    hipFree(d_idata);
    hipFree(d_odata[0]);
    hipFree(d_odata[1]);
    hipFree(d_odata[2]);
}
