#include <stdio.h>
#include "reduction_kernel.cu"

//***********************************************************************
//! The Driver
// The number of threads must be not less than wA * wB
//***********************************************************************

int main() {
  int input[NUM * 2];
  int output[1];
  klee_make_symbolic(input, sizeof(input), "input");

  int *dinput, *doutput;
  hipMalloc((void**)&dinput, sizeof(int)*NUM*2);
  hipMalloc((void**)&doutput, sizeof(int));
  hipMemcpy(dinput, input, sizeof(int)*NUM*2, hipMemcpyHostToDevice);

#if defined RED0
  FUNC(reduce0)<<<1, NUM>>>(dinput, doutput, NUM);
#elif defined RED1 
  FUNC(reduce1)<<<1, NUM>>>(dinput, doutput, NUM);
#elif defined RED2
  FUNC(reduce2)<<<1, NUM>>>(dinput, doutput, NUM);
#elif defined RED3
  FUNC(reduce3)<<<1, NUM>>>(dinput, doutput, NUM);
#elif defined RED4
  FUNC(reduce4)<<<1, NUM>>>(dinput, doutput, NUM, NUM);
#elif defined RED5
  FUNC(reduce5)<<<1, NUM>>>(dinput, doutput, NUM, NUM);
#else
  FUNC(reduce6)<<<1, NUM>>>(dinput, doutput, NUM, NUM, false);
#endif

  hipMemcpy(output, doutput, sizeof(int), hipMemcpyDeviceToHost);

#ifndef _SYM
  // post-condition
  int sum = 0;
  for (int i = 0; i < NUM; i++) 
    sum += input[i];
  if (sum != output[0]) {
    printf("Incorrect: sum = %d, output[0] = %d \n", sum, output[0]); 
    return 0;
  }
  return 1;
#endif
}
