#include "hip/hip_runtime.h"
#include "stdio.h"

#define    NUM  4 

__shared__ int shared[NUM];

__device__ inline void swap(int & a, int & b) {
  // Alternative swap doesn't use a temporary register:
  // a ^= b;
  // b ^= a;
  // a ^= b;
  
  int tmp = a;
  a = b;
  b = tmp;
}

__global__ void BitonicKernel(int * values)
{
  unsigned int tid = threadIdx.x;
  // Copy input to shared mem.  
  shared[tid] = values[tid];
  printf("tid: %d, blockDim: %d\n", tid, blockDim.x);
  __syncthreads();

  // Parallel bitonic sort.
  for (unsigned int k = 2; k <= blockDim.x; k *= 2) {
    for (unsigned int j = k / 2; j>0; j /= 2) {
      unsigned int ixj = tid ^ j;
      if (ixj > tid) {
	if ((tid & k) == 0) {
	  if (shared[tid] > shared[ixj])
	    swap(shared[tid], shared[ixj]);
	}
	else {
	  if (shared[tid] < shared[ixj])
	    swap(shared[tid], shared[ixj]);
	}
      }
      __syncthreads();
    }
  } //end sort
  
  // Write result.
  values[tid] = shared[tid];
}


int main() {

  int *values = (int *)malloc(sizeof(int) * NUM); 
  // the following is equivalent to calling the kernel using <<<...>>>(BitonicKernel)
  klee_make_symbolic(values, sizeof(int)*NUM, "values");

  int *dvalues;
  hipMalloc((void **)&dvalues, sizeof(int) * NUM);
  hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice);
  
  BitonicKernel<<<1, NUM>>>(dvalues);

#ifndef _SYM
  // here blockDim.x should be NUM; we use this hack for convenience
  for (int i = 1; i < NUM; i++) {
    if (dvalues[i] < dvalues[i-1]) {
      printf("The sorting algorithm is incorrect since values[%d] < values[%d]!\n", i, i-1);
      return 1;
    }
  }
#endif

  hipFree(dvalues);
  free(values);

  return 0;
}
