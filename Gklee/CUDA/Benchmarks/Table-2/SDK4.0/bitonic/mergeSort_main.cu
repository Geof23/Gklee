/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include "mergeSort_common.h"
#include "mergeSort.cu"
#include "mergeSort_validate.cpp"



////////////////////////////////////////////////////////////////////////////////
// Test driver
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv){
    uint 
        *h_SrcKey, *h_SrcVal, *h_DstKey, *h_DstVal;
    uint 
        *d_SrcKey, *d_SrcVal, *d_BufKey, *d_BufVal, *d_DstKey, *d_DstVal;
    uint hTimer;

    //const uint   N = 4 * 1048576;
    const uint   N = 4 * 1024;
    const uint DIR = 1;

    const uint numValues = 65536;


    printf("Allocating and initializing host arrays...\n\n");
        h_SrcKey = (uint *)malloc(N * sizeof(uint));
        h_SrcVal = (uint *)malloc(N * sizeof(uint));
        h_DstKey = (uint *)malloc(N * sizeof(uint));
        h_DstVal = (uint *)malloc(N * sizeof(uint));

        srand(2009);
        for(uint i = 0; i < N; i++)
            h_SrcKey[i] = rand() % numValues;
        fillValues(h_SrcVal, N);

    printf("Allocating and initializing CUDA arrays...\n\n");
        hipMalloc((void **)&d_DstKey, N * sizeof(uint));
        hipMalloc((void **)&d_DstVal, N * sizeof(uint));
        hipMalloc((void **)&d_BufKey, N * sizeof(uint));
        hipMalloc((void **)&d_BufVal, N * sizeof(uint));
        hipMalloc((void **)&d_SrcKey, N * sizeof(uint));
        hipMalloc((void **)&d_SrcVal, N * sizeof(uint));
        hipMemcpy(d_SrcKey, h_SrcKey, N * sizeof(uint), hipMemcpyHostToDevice);
        hipMemcpy(d_SrcVal, h_SrcVal, N * sizeof(uint), hipMemcpyHostToDevice);

    printf("Initializing GPU merge sort...\n");
        initMergeSort();

    printf("Running GPU merge sort...\n");
        hipDeviceSynchronize();
            mergeSort(
                d_DstKey,
                d_DstVal,
                d_BufKey,
                d_BufVal,
                d_SrcKey,
                d_SrcVal,
                N,
                DIR
            );
        hipDeviceSynchronize();

    printf("Reading back GPU merge sort results...\n");
        hipMemcpy(h_DstKey, d_DstKey, N * sizeof(uint), hipMemcpyDeviceToHost);
        hipMemcpy(h_DstVal, d_DstVal, N * sizeof(uint), hipMemcpyDeviceToHost);

    printf("Inspecting the results...\n");
        uint keysFlag = validateSortedKeys(
            h_DstKey,
            h_SrcKey,
            1,
            N,
            numValues,
            DIR
        );

        uint valuesFlag = validateSortedValues(
            h_DstKey,
            h_DstVal,
            h_SrcKey,
            1,
            N
        );

    printf( (keysFlag && valuesFlag) ? "TEST PASSED\n" : "TEST FAILED\n");

    printf("Shutting down...\n");
        closeMergeSort();
        hipFree(d_SrcVal);
        hipFree(d_SrcKey);
        hipFree(d_BufVal);
        hipFree(d_BufKey);
        hipFree(d_DstVal);
        hipFree(d_DstKey);
        free(h_DstVal);
        free(h_DstKey);
        free(h_SrcVal);
        free(h_SrcKey);
        hipDeviceReset();
}
