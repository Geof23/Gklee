#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM 4

__global__
void clzKernel(int *uA, int *uB) {
  unsigned tid = threadIdx.x;
  uB[tid] = __clz(uA[tid]); 
}

__global__
void ffsKernel(int *uA, int *uB) {
  unsigned tid = threadIdx.x;
  uB[tid] = __ffs(uA[tid]); 
}

__global__
void popcKernel(unsigned *uA, unsigned *uB) {
  unsigned tid = threadIdx.x;
  uB[tid] = __popc(uA[tid]); 
}

__global__
void brevKernel(unsigned *uA, unsigned *uB) {
  unsigned tid = threadIdx.x;
  uB[tid] = __brev(uA[tid]); 
}

__global__
void bytePermKernel(unsigned *uA, unsigned *uB, 
                    unsigned *uC, unsigned *uD) {
  unsigned tid = threadIdx.x;
  uD[tid] = __byte_perm(uA[tid], uB[tid], uC[tid]); 
}

int main(int argv, char **argc) {
  int hA[NUM] = {1, 2, 3, 4};
  int hB[NUM] = {1, 2, 3, 4};

  int *dA, *dB;
  hipMalloc((void**)&dA, sizeof(int)*NUM);
  hipMalloc((void**)&dB, sizeof(int)*NUM);

  hipMemcpy(dA, hA, sizeof(int)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, sizeof(int)*NUM, hipMemcpyHostToDevice);

  clzKernel<<<1, NUM>>>(dA, dB);

  hipMemcpy(hB, dB, sizeof(int)*NUM, hipMemcpyDeviceToHost);
 
  printf("After clz intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("hB[%u]: %d\n", i, hB[i]);
  } 

  ffsKernel<<<1, NUM>>>(dA, dB);

  hipMemcpy(hB, dB, sizeof(int)*NUM, hipMemcpyDeviceToHost);
 
  printf("After ffs intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("hB[%u]: %d\n", i, hB[i]);
  } 

  unsigned huA[NUM] = {1, 2, 3, 4};
  unsigned huB[NUM] = {1, 2, 3, 4};
  unsigned huC[NUM] = {1, 2, 3, 4};
  unsigned huD[NUM] = {1, 2, 3, 4};

  unsigned *duA, *duB, *duC, *duD;

  hipMalloc((void**)&duA, sizeof(unsigned)*NUM);
  hipMalloc((void**)&duB, sizeof(unsigned)*NUM);
  hipMalloc((void**)&duC, sizeof(unsigned)*NUM);
  hipMalloc((void**)&duD, sizeof(unsigned)*NUM);

  hipMemcpy(duA, huA, sizeof(unsigned)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(duB, huB, sizeof(unsigned)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(duC, huC, sizeof(unsigned)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(duD, huD, sizeof(unsigned)*NUM, hipMemcpyHostToDevice);

  popcKernel<<<1, NUM>>>(duA, duB); 

  hipMemcpy(huB, duB, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
 
  printf("After popc intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("huB[%u]: %u\n", i, huB[i]);
  } 

  brevKernel<<<1, NUM>>>(duA, duB); 

  hipMemcpy(huB, duB, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
 
  printf("After brev intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("huB[%u]: %u\n", i, huB[i]);
  } 

  bytePermKernel<<<1, NUM>>>(duA, duB, duC, duD); 

  hipMemcpy(huD, duD, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
 
  printf("After bytePerm intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("huD[%u]: %u\n", i, huD[i]);
  } 

  hipFree(dA);
  hipFree(dB);

  hipFree(duA);
  hipFree(duB);
  hipFree(duC);
  hipFree(duD);
}
