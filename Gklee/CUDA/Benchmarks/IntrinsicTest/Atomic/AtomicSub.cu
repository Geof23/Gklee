#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM 4

__global__
void atomicIntSubKernel(int *uA, int *uB, int *uC) {
  unsigned tid = threadIdx.x;
  uC[tid] = atomicSub(uA+tid, uB[tid]); 
}

__global__
void atomicUnsignedSubKernel(unsigned *uA, unsigned *uB, unsigned *uC) {
  unsigned tid = threadIdx.x;
  uC[tid] = atomicSub(uA+tid, uB[tid]); 
}

__global__
void atomicIntExchKernel(int *uA, int *uB, int *uC) {
  unsigned tid = threadIdx.x;
  uC[tid] = atomicExch(uA+tid, uB[tid]); 
}

__global__
void atomicUnsignedExchKernel(unsigned *uA, unsigned *uB, unsigned *uC) {
  unsigned tid = threadIdx.x;
  uC[tid] = atomicExch(uA+tid, uB[tid]); 
}

__global__
void atomicULLExchKernel(unsigned long long int *uA, unsigned long long int *uB, 
                         unsigned long long int *uC) {
  unsigned tid = threadIdx.x;
  uC[tid] = atomicExch(uA+tid, uB[tid]); 
}

__global__
void atomicFloatExchKernel(float *fA, float *fB, float *fC) {
  unsigned tid = threadIdx.x;
  fC[tid] = atomicExch(fA+tid, fB[tid]); 
}

int main(int argv, char **argc) {
  int hiA[NUM] = {1, 2, 3, 4}; 
  int hiB[NUM] = {1, 2, 3, 4}; 
  int hiC[NUM] = {0, 0, 0, 0};

  unsigned hA[NUM] = {1, 2, 3, 4}; 
  unsigned hB[NUM] = {1, 2, 3, 4}; 
  unsigned hC[NUM] = {0, 0, 0, 0};

  unsigned long long int hullA[NUM] = {1, 2, 3, 4}; 
  unsigned long long int hullB[NUM] = {1, 2, 3, 4}; 
  unsigned long long int hullC[NUM] = {0, 0, 0, 0};

  float hfA[NUM] = {1.0, 2.0, 3.0, 4.0}; 
  float hfB[NUM] = {1.0, 2.0, 3.0, 4.0}; 
  float hfC[NUM] = {0.0, 0.0, 0.0, 0.0}; 

  //klee_make_symbolic(hiA, sizeof(int)*NUM, "hiA");
  //klee_make_symbolic(hiB, sizeof(int)*NUM, "hiB");

  //klee_make_symbolic(hA, sizeof(unsigned)*NUM, "hA");
  //klee_make_symbolic(hB, sizeof(unsigned)*NUM, "hB");

  //klee_make_symbolic(hullA, sizeof(unsigned long long int)*NUM, "hullA");
  //klee_make_symbolic(hullB, sizeof(unsigned long long int)*NUM, "hullB");

  //klee_make_symbolic(hfA, sizeof(float)*NUM, "hfA");
  //klee_make_symbolic(hfB, sizeof(float)*NUM, "hfB");

  int *diA, *diB, *diC; 
  hipMalloc((void**)&diA, sizeof(int)*NUM);
  hipMalloc((void**)&diB, sizeof(int)*NUM);
  hipMalloc((void**)&diC, sizeof(int)*NUM);

  unsigned *dA, *dB, *dC; 
  hipMalloc((void**)&dA, sizeof(unsigned)*NUM);
  hipMalloc((void**)&dB, sizeof(unsigned)*NUM);
  hipMalloc((void**)&dC, sizeof(unsigned)*NUM);

  unsigned long long int *dullA, *dullB, *dullC; 
  hipMalloc((void**)&dullA, sizeof(unsigned long long int)*NUM);
  hipMalloc((void**)&dullB, sizeof(unsigned long long int)*NUM);
  hipMalloc((void**)&dullC, sizeof(unsigned long long int)*NUM);

  float *dfA, *dfB, *dfC; 
  hipMalloc((void**)&dfA, sizeof(float)*NUM);
  hipMalloc((void**)&dfB, sizeof(float)*NUM);
  hipMalloc((void**)&dfC, sizeof(float)*NUM);

  hipMemcpy(diA, hiA, sizeof(int)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(diB, hiB, sizeof(int)*NUM, hipMemcpyHostToDevice);

  hipMemcpy(dA, hA, sizeof(unsigned)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, sizeof(unsigned)*NUM, hipMemcpyHostToDevice);

  hipMemcpy(dullA, hullA, sizeof(unsigned long long int)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(dullB, hullB, sizeof(unsigned long long int)*NUM, hipMemcpyHostToDevice);

  hipMemcpy(dfA, hfA, sizeof(float)*NUM, hipMemcpyHostToDevice);
  hipMemcpy(dfB, hfB, sizeof(float)*NUM, hipMemcpyHostToDevice);

  atomicIntSubKernel<<<1, NUM>>>(diA, diB, diC);

  atomicUnsignedSubKernel<<<1, NUM>>>(dA, dB, dC); 

  hipMemcpy(hiA, diA, sizeof(int)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hiB, diB, sizeof(int)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hiC, diC, sizeof(int)*NUM, hipMemcpyDeviceToHost);
 
  hipMemcpy(hA, dA, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hB, dB, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hC, dC, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
 
  printf("After atomic Sub intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("hiA[%u]: %d\n", i, hiA[i]);
    printf("hiB[%u]: %d\n", i, hiB[i]);
    printf("hiC[%u]: %d\n", i, hiC[i]);
    printf("****** \n");
    printf("hA[%u]: %u\n", i, hA[i]);
    printf("hB[%u]: %u\n", i, hB[i]);
    printf("hC[%u]: %u\n", i, hC[i]);
  } 

  atomicIntExchKernel<<<1, NUM>>>(diA, diB, diC); 

  atomicUnsignedExchKernel<<<1, NUM>>>(dA, dB, dC); 

  atomicULLExchKernel<<<1, NUM>>>(dullA, dullB, dullC); 

  atomicFloatExchKernel<<<1, NUM>>>(dfA, dfB, dfC); 

  hipMemcpy(hiA, diA, sizeof(int)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hiB, diB, sizeof(int)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hiC, diC, sizeof(int)*NUM, hipMemcpyDeviceToHost);
 
  hipMemcpy(hA, dA, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hB, dB, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hC, dC, sizeof(unsigned)*NUM, hipMemcpyDeviceToHost);
 
  hipMemcpy(hullA, dullA, sizeof(unsigned long long int)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hullB, dullB, sizeof(unsigned long long int)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hullC, dullC, sizeof(unsigned long long int)*NUM, hipMemcpyDeviceToHost);
 
  hipMemcpy(hfA, dfA, sizeof(float)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hfB, dfB, sizeof(float)*NUM, hipMemcpyDeviceToHost);
  hipMemcpy(hfC, dfC, sizeof(float)*NUM, hipMemcpyDeviceToHost);
 
  printf("After atomic Exch intrinsic: \n");
  for (unsigned i = 0; i < NUM; i++) {
    printf("hiA[%u]: %d\n", i, hiA[i]);
    printf("hiB[%u]: %d\n", i, hiB[i]);
    printf("hiC[%u]: %d\n", i, hiC[i]);
    printf("****** \n");
    printf("hA[%u]: %u\n", i, hA[i]);
    printf("hB[%u]: %u\n", i, hB[i]);
    printf("hC[%u]: %u\n", i, hC[i]);
    printf("****** \n");
    printf("hullA[%u]: %llu\n", i, hullA[i]);
    printf("hullB[%u]: %llu\n", i, hullB[i]);
    printf("hullC[%u]: %llu\n", i, hullC[i]);
    printf("****** \n");
    printf("hfA[%u]: %f\n", i, hfA[i]);
    printf("hfB[%u]: %f\n", i, hfB[i]);
    printf("hfC[%u]: %f\n", i, hfC[i]);
    printf("------ \n");
  } 

  hipFree(diA);
  hipFree(diB);
  hipFree(diC);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dullA);
  hipFree(dullB);
  hipFree(dullC);
  hipFree(dfA);
  hipFree(dfB);
  hipFree(dfC);
}
