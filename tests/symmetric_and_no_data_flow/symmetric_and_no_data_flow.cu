
#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS 32

// from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

__global__ void device_global(unsigned int *input_array, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;

  // all threads write their index into the array
  input_array[my_index] = my_index;

  __syncthreads();

  // all threads write to the array from values in th array
  // written by the second neighbor thread, causing a read-write race
  // the mod is so the last even thread reads from the first
  // even thread's index
  //
  // since the value never goes into an indexing position there is no
  // flow
  input_array[my_index] = input_array[(my_index+2) % THREADS];
}

int main(void) {
  // how big our array for interfacing with the GPU will be
  int num_elements = THREADS;
  int num_bytes = sizeof(unsigned int) * num_elements;
    
  // pointers for the interfacing arrays
  unsigned int *host_array = 0;
  unsigned int *device_array = 0;
 
  // malloc for host and device
  host_array = (unsigned int*) malloc(num_bytes);
  hipMalloc((void **) &device_array, num_bytes);

  // check the mallocs
  if (host_array == 0) {
    printf("Unable to allocate memory on host");
    return 1;
  }

  if (device_array == 0) {
    printf("Unable to allocate memory on device");
    return 1;
  }

  // set host array values
  for (int i = 0; i<num_elements; i++) {
    host_array[i] = 777777;
  }

  // copy them to the GPU
  hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);

  // define block and grid sizes
  int block_size = THREADS;
  int grid_size = (num_elements + block_size - 1) / block_size;

  // run GPU code
  device_global<<<grid_size, block_size>>>(device_array, num_elements);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // copy output to host
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print any information
  for (int i=0; i<num_elements; i++) {
    printf("%6u, ", host_array[i]);
    if (i%10 == 9) {
      printf(" \n");
    }
  }
  printf("\n");

  // free memory
  free(host_array);
  hipFree(device_array);
}
