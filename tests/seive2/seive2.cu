
#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS 32

// from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

__global__ void device_global(unsigned int *input_array, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i=0; i<THREADS; i++) {
    if (i<my_index) {
      if (((my_index+2) % input_array[i]) == 0) {
	input_array[my_index] = 0;
      }
    }
    __syncthreads();
  }
}

int main(void) {
  // how big our array for interfacing with the GPU will be
  int num_elements = THREADS;
  int num_bytes = sizeof(unsigned int) * num_elements;
    
  // pointers for the interfacing arrays
  unsigned int *host_array = 0;
  unsigned int *device_array = 0;
 
  // malloc for host and device
  host_array = (unsigned int*) malloc(num_bytes);
  hipMalloc((void **) &device_array, num_bytes);

  // check the mallocs
  if (host_array == 0) {
    printf("Unable to allocate memory on host");
    return 1;
  }

  if (device_array == 0) {
    printf("Unable to allocate memory on device");
    return 1;
  }

  // set host array values
  for (int i = 0; i<num_elements; i++) {
    host_array[i] = i+2;
  }

  // copy them to the GPU
  hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);

  // define block and grid sizes
  int block_size = THREADS/2;
  int grid_size = (num_elements + block_size - 1) / block_size;

  // run GPU code
  device_global<<<grid_size, block_size>>>(device_array, num_elements);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  // copy output to host
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print any information
  int j = 0;
  for (int i=0; i<num_elements; i++) {
    if (host_array[i] != 0) {
      printf("%03u, ", host_array[i]);
      if (j%10 == 9) {
	printf(" \n");
      }
      j += 1;
    }
  }
  printf("\n");
  // free memory
  free(host_array);
  hipFree(device_array);
}
