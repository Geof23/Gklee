
#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS 64

// from http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

__global__ void device_global(unsigned int *input_array, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;

  // all threads write a value to the array
  input_array[my_index] = my_index - (my_index%2);

  __syncthreads(); // all initial values are written

  // all threads grab a value from the array
  // we know this will always be in bounds
  int new_index = input_array[my_index];
  
  __syncthreads(); // all values are read

  // use the values to write to the array, a write-write race
  input_array[new_index] = my_index;
}

int main(void) {
  // how big our array for interfacing with the GPU will be
  int num_elements = THREADS;
  int num_bytes = sizeof(unsigned int) * num_elements;
    
  // pointers for the interfacing arrays
  unsigned int *host_array = 0;
  unsigned int *device_array = 0;
 
  // malloc for host and device
  host_array = (unsigned int*) malloc(num_bytes);
  hipMalloc((void **) &device_array, num_bytes);

  // check the mallocs
  if (host_array == 0) {
    printf("Unable to allocate memory on host");
    return 1;
  }

  if (device_array == 0) {
    printf("Unable to allocate memory on device");
    return 1;
  }

  // set host array values
  for (int i = 0; i<num_elements; i++) {
    host_array[i] = 777777;
  }

  // copy them to the GPU
  hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);

  // define block and grid sizes
  int block_size = THREADS;
  int grid_size = (num_elements + block_size - 1) / block_size;

  // run GPU code
  device_global<<<grid_size, block_size>>>(device_array, num_elements);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // copy output to host
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print any information
  for (int i=0; i<num_elements; i++) {
    printf("%6u, ", host_array[i]);
    if (i%10 == 9) {
      printf(" \n");
    }
  }
  printf("\n");

  // free memory
  free(host_array);
  hipFree(device_array);
}
