
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 50
#define B 2
#define T 32

__global__ void dl(int* in)
{
  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid < N)
  {
    if(in[tid] % 2 == 0)
      in[tid]++;

    __syncthreads(); // ouch

    int sum = in[tid];
    if(tid > 0)
      sum += in[tid-1];
    if(tid < N - 1)
      sum += in[tid+1];
    in[tid] = sum / 3;
  }
}

int main()
{
  int* in = (int*) malloc(N*sizeof(int));
  
  for(int i = 0; i < N; i++)
    in[i] = i;
  
  int* din;
  hipMalloc((void**)&din, N*sizeof(int));
  hipMemcpy(din, in, N*sizeof(int), hipMemcpyHostToDevice);

  dl<<<B,T>>>(din);

  hipMemcpy(in, din, N*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
    printf("%d ", in[i]);
  printf("\n");
  free(in); hipFree(din);
}