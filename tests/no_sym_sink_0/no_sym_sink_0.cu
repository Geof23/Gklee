
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 8

__global__ void k(int* in)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  // Gkleep reports out of bound pointer...
  // Causes gkleep to use one flow per thread.
  int write_val = N-tid-1;

  for(int i = 0; i <= tid; i++)
    in[tid] = write_val;
  
  __syncthreads();

  if(tid % 2 == 1)
    in[tid] = in[in[tid]];  
}

int main()
{
  int* din;
  hipMalloc((void**) &din, sizeof(int)*N);
  k<<<1,N>>>(din);

  int in[N];
  hipMemcpy(&in, din, sizeof(int)*N, hipMemcpyDeviceToHost);
  for(int i = 0; i < N; i++)
    printf("%4d, ", in[i]);
  printf("\n");

  hipFree(din);
  return 0;
}