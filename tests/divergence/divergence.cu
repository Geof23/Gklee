
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 50
#define T 128
#define B 2

__global__ void div(int* in, int* out)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid < N)
  {
    if(tid % 2 == 0)
      out[tid] = in[tid] - 1;
    else
      out[tid] = in[tid] + 1;
  }
}

int main()
{
  int* in = (int*) malloc(N*sizeof(int));

  for(int i = 0; i < N; i++)
    in[i] = i;

  int* din, *dout;
  hipMalloc((void**)&din, N*sizeof(int));
  hipMalloc((void**)&dout,N*sizeof(int));
  
  hipMemcpy(din, in, N*sizeof(int), hipMemcpyHostToDevice);

  div<<<B,T>>>(din,dout);

  hipMemcpy(in, dout, N*sizeof(int), hipMemcpyDeviceToHost);
  for(int i = 0; i < N; i++)
    printf("%d ", in[i]);
  printf("\n");

  free(in); hipFree(din); hipFree(dout);
}