
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void iwarp(int* out)
{
  volatile int* vout = out;
  *vout = threadIdx.x;
}

int main()
{
  int* din;
  hipMalloc((void**)&din, sizeof(int));
  int in = 0;
  hipMemcpy(din, &in, sizeof(int), hipMemcpyHostToDevice);
  iwarp<<<1,16>>>(din);
  int output;
  hipMemcpy(&output, din, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", output);
}