/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include <hip/hip_runtime.h>
#include <cstdio>

//#include "../common/book.h"

#define N   10

__global__ void add( int *a, int *b, int *c ) {
  int tid = blockIdx.x;    // this thread handles the data at its thread id
  if (tid < N)
    c[tid] = a[tid] + b[tid];
}

int main( void ) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  // allocate the memory on the GPU
  hipMalloc( (void**)&dev_a, N * sizeof(int) ) ;
  hipMalloc( (void**)&dev_b, N * sizeof(int) ) ;
  hipMalloc( (void**)&dev_c, N * sizeof(int) ) ;

  // fill the arrays 'a' and 'b' on the CPU
  for (int i=0; i<N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  // copy the arrays 'a' and 'b' to the GPU
  hipMemcpy( dev_a, a, N * sizeof(int),
	      hipMemcpyHostToDevice ) ;
  hipMemcpy( dev_b, b, N * sizeof(int),
	      hipMemcpyHostToDevice );

  add<<<N,1>>>( dev_a, dev_b, dev_c );

  // copy the array 'c' back from the GPU to the CPU
  hipMemcpy( c, dev_c, N * sizeof(int),
	      hipMemcpyDeviceToHost ) ;

  // display the results
  for (int i=0; i<N; i++) {
    printf( "%d + %d = %d\n", a[i], b[i], c[i] );
  }

  // free the memory allocated on the GPU
 hipFree( dev_a ) ;
 hipFree( dev_b ) ;
 hipFree( dev_c ) ;

  return 0;
}
