
#include <hip/hip_runtime.h>
#include <cstdio>

#define N 64
#define B 1
#define T 64

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
     fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
     if (abort) exit(code);
   }
}

__global__ void dl(int* in)
{
  
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // The warps in this block take different paths; the synctreads calls
  // will cause a deadlock.
  if(tid > 31)
  {
    if(in[tid] % 2 == 0)
      in[tid]++;

    __syncthreads();

  }
  else {
    if(in[tid] % 2 == 1)
      in[tid]--;
    
    __syncthreads();
  }
/*  int sum = in[tid];
  if(tid > 0)
    sum += in[tid-1];
  if(tid < N - 1)
      sum += in[tid+1];
      in[tid] = sum / 3; */
}

int main()
{
  int* in = (int*) malloc(N*sizeof(int));
  
  for(int i = 0; i < N; i++)
    in[i] = i;
  
  int* din;
  gpuErrchk(hipMalloc((void**)&din, N*sizeof(int)));
  gpuErrchk(hipMemcpy(din, in, N*sizeof(int), hipMemcpyHostToDevice));

  dl<<<B,T>>>(din);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(in, din, N*sizeof(int), hipMemcpyDeviceToHost));

  for(int i = 0; i < N; i++)
    printf("%d ", in[i]);
  printf("\n");
  free(in); hipFree(din);
}