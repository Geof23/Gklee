// Exhibits a bank conflict.
// Gklee and Gkleep both detect this.


#include <hip/hip_runtime.h>
#include <cstdio>

#define N 32

__global__ void bc(char* in, char* out)
{
  __shared__ int smem[512];

  int tid = threadIdx.x;
  
  smem[tid*2]=in[tid];
  __syncthreads();
  smem[tid*4]=in[tid];
  __syncthreads();
  smem[tid*8]=in[tid];
  __syncthreads();

  int x = smem[tid * 2]; // 2-way bank conflicts
  int y = smem[tid * 4]; // 4-way bank conflicts
  int z = smem[tid * 8]; // 8-way bank conflicts
  
  int m = max(max(x,y),z);
  out[tid] = m;
}

int main()
{
  char* in = (char*) malloc(N*sizeof(char));
  for(int i = 0; i < N; i++)
    in[i] = i;
  
  char* din, * dout;
  hipMalloc((void**) &din, N*sizeof(char));
  hipMalloc((void**) &dout, N*sizeof(char));
  
  hipMemcpy(din, in, N*sizeof(char), hipMemcpyHostToDevice);
  
  bc<<<1,N>>>(din,dout);
  
  hipMemcpy(in, dout, N*sizeof(char), hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
    printf("%d ", in[i]);
  printf("\n");
  free(in); hipFree(din); hipFree(dout);
}