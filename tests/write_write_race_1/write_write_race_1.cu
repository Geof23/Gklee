
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void device_global(unsigned int *input_array, int num_elements) {
  int my_index = blockIdx.x * blockDim.x + threadIdx.x;
  input_array[0] = my_index;
  }


int main(void) {
  // how big our array for interfacing with the GPU will be
  int num_elements = 1;
  int num_bytes = sizeof(unsigned int) * num_elements;
    
  // pointers for the interfacing arrays
  unsigned int *host_array = 0;
  unsigned int *device_array = 0;
 
  // malloc for host and device
  host_array = (unsigned int*) malloc(num_bytes);
  hipMalloc((void **) &device_array, num_bytes);

  // check the mallocs
  if (host_array == 0) {
    printf("Unable to allocate memory on host");
    return 1;
  }

  if (device_array == 0) {
    printf("Unable to allocate memory on device");
    return 1;
  }

  // set host array values
  for (int i = 0; i<num_elements; i++) {
    host_array[i] = 0;
  }

  // copy them to the GPU
  hipMemcpy(device_array, host_array, num_bytes, hipMemcpyHostToDevice);

  // define block and grid sizes
  int block_size = 32;
  int grid_size = (num_elements + block_size - 1) / block_size;

  // run GPU code
  device_global<<<grid_size, block_size>>>(device_array, num_elements);

  // copy output to host
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print any information
  printf("host_array[0] = %u \n", host_array[0]);

  // free memory
  free(host_array);
  hipFree(device_array);
}