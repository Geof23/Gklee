// Tests executing two kernels, with host code between kernel launches.


#include <hip/hip_runtime.h>
#include <cstdio>

#define N 100

__global__ void kernel1(int* in, int* out)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if(idx < N)
    out[idx] = in[idx] + 1;
}

__global__ void kernel2(int*in, int*out)
{
  int idx = threadIdx.x + blockDim.x*blockIdx.x;
  if(idx < N)
    out[idx] = in[idx]*in[idx];
}

int main()
{
  int* in = (int*) malloc(N*sizeof(int));
  for(int i = 0; i < N; i++)
    in[i] = i;
  int* din;
  int* dout;
  hipMalloc((void**)&din, N*sizeof(int));
  hipMalloc((void**)&dout, N*sizeof(int));

  hipMemcpy(din, in, N*sizeof(int), hipMemcpyHostToDevice);
  kernel1<<<1,N>>>(din, dout);

  hipMemcpy(in, dout, N*sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++)
    in[i]--;

  kernel2<<<1,N>>>(din, dout);
  
  hipMemcpy(in, dout, N*sizeof(int), hipMemcpyDeviceToHost);
  for(int i = 1; i < N; i++)
    {
      in[i] = in[i]/i;
      printf("%d ", in[i]);
    }
  printf("\n");
}